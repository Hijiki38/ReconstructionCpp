#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <time.h>

// GPU�Ōv�Z����ۂ̊֐�
__global__ void gpu_function(float* d_x, float* d_y)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	d_y[i] = sin(d_x[i]) * sin(d_x[i]) + cos(d_x[i]) * cos(d_x[i]);
}

// CPU�Ōv�Z����ۂ̊֐�
void cpu_function(int n, float* x, float* y)
{
	for (int i = 0; i < n; i++) {
		y[i] = sin(x[i]) * sin(x[i]) + cos(x[i]) * cos(x[i]);
	}
}

// main function
int cudamain(void)
{
	bool GPU = true;

	int N = 1000000;
	float* host_x, * host_y, * dev_x, * dev_y;

	// CPU���̗̈�m��
	host_x = (float*)malloc(N * sizeof(float));
	host_y = (float*)malloc(N * sizeof(float));

	// �����l����͂���
	for (int i = 0; i < N; i++) {
		host_x[i] = rand();
	}

	int start = clock();

	if (GPU == true) {

		// �f�o�C�X(GPU)���̗̈�m��
		hipMalloc(&dev_x, N * sizeof(float));
		hipMalloc(&dev_y, N * sizeof(float));

		// CPU��GPU�̃f�[�^�R�s�[
		hipMemcpy(dev_x, host_x, N * sizeof(float), hipMemcpyHostToDevice);

		// GPU�Ōv�Z
		gpu_function << <(N + 255) / 256, 256 >> > (dev_x, dev_y);

		// GPU��CPU�̃f�[�^�R�s�[
		hipMemcpy(host_y, dev_y, N * sizeof(float), hipMemcpyDeviceToHost);

	}
	else {
		// CPU�Ōv�Z
		cpu_function(N, host_x, host_y);
	}

	int end = clock();

	// �v�Z���������s���Ă��邩�m�F
	float sum = 0.0f;
	for (int j = 0; j < N; j++) {
		sum += host_y[j];
	}
	std::cout << sum << std::endl;

	// �Ō�Ɍv�Z���Ԃ�\��
	std::cout << end - start << "[ms]" << std::endl;

	return 0;
}