#include "hip/hip_runtime.h"
#include "CalcSysmat.cuh"


#define CHECK(call)\
{\
    const hipError_t error = call;\
    if (error != hipSuccess)\
        {\
            printf("Error: %s:%d",__FILE__,__LINE__);\
            printf("code:%d, reason: %s\n", error, hipGetErrorString(error));\
            exit(1);\
        }\
}

namespace Reconstruction {

	__global__ void calc_coeff_cbct(float* result, const int nd, const int center,
		const int w, const int theta, const int sdd, const int rotcount) {

		unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
		unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
		unsigned int idx = iy * nd + ix;

		float relx = ix;
		float rely = iy;
		float tmp_x;
		float offset_detector, phi, intercept;

		float la1, la2, lb1, lb2, sa, sb;
		float a = 0.5; // = pixsize / 2
		float tan_angle;
		//float tan_delta = a / sdd;
		float tan_delta = 0;

		if (ix < nd && iy < nd) {
			for (int i = 0; i < rotcount; i++) {
				tmp_x = relx;
				relx = 2 * center - rely - 1;
				rely = tmp_x;
			}
			//float point::get_relative(float _num) { return _num - center + 0.5; }

			relx = relx - center + 0.5;
			rely = center - rely - 0.5;

			offset_detector = (nd - w - 1 - center + 0.5) / cosf(theta); //offset of the detector
			//phi = atan2f(offset_detector, sdd);
			phi = 0;

			tan_angle = tanf(theta + phi);
			intercept = offset_detector - rely + relx * tan_angle;

			la1 = a - (-a * (tan_angle + tan_delta) + intercept + a / sqrt(1 / (1 + (tan_angle + tan_delta) * (tan_angle + tan_delta))));
			la2 = a - (a * (tan_angle + tan_delta) + intercept + a / sqrt(1 / (1 + (tan_angle + tan_delta) * (tan_angle + tan_delta))));
			lb1 = a + (-a * (tan_angle - tan_delta) + intercept - a / sqrt(1 / (1 + (tan_angle - tan_delta) * (tan_angle - tan_delta))));
			lb2 = a + (a * (tan_angle - tan_delta) + intercept - a / sqrt(1 / (1 + (tan_angle - tan_delta) * (tan_angle - tan_delta))));

			if (la1 < 0) {
				if (la2 < 0) {
					sa = 0;
				}
				else {
					sa = a * la2 * la2 / (-la1 + la2);
				}
			}
			else if (la1 < 2 * a) {
				if (la2 < 0) {
					sa = a * la1 * la1 / (la1 - la2);
				}
				else if (la2 < 2 * a) {
					sa = a * (la1 + la2);
				}
			}
			else {
				if (la2 < 2 * a) {
					sa = a * (la1 + la2) - (la1 - 2 * a) * (la1 - 2 * a) / (2 * (la1 - la2));
				}
				else {
					sa = (2 * a) * (2 * a);
				}
			}

			if (lb1 < 0) {
				if (lb2 < 0) {
					sb = 0;
				}
				else {
					sb = a * lb2 * lb2 / (-lb1 + lb2);
				}
			}
			else if (lb1 < 2 * a) {
				if (lb2 < 0) {
					sb = a * lb1 * lb1 / (lb1 -lb2);
				}
				else if (lb2 < 2 * a) {
					sb = a * (lb1 + lb2);
				}
			}
			else {
				if (lb2 < 2 * a) {
					sb = a * (lb1 + lb2) - (lb1 - 2 * a) * (lb1 - 2 * a) / (2 * (lb1 - lb2));
				}
				else {
					sb = (2 * a) * (2 * a);
				}
			}

			result[idx] = (2 * a) * (2 * a) - (sa + sb);
		}
	}

	void calc_sysmat(float* result, const int nd, const int center,
		const int w, const int theta, const int sdd, const int rotcount) {

		int dev = 0;
		hipDeviceProp_t deviceprop;
		CHECK(hipGetDeviceProperties(&deviceprop, dev));
		CHECK(hipSetDevice(dev));

		int nxy = nd * nd;
		int nBytes = nxy * sizeof(float);

		float* d_res;
		CHECK(hipMalloc((void**)&d_res, nBytes));

		CHECK(hipMemcpy(d_res, result, nBytes, hipMemcpyHostToDevice));

		int dimx = 32;
		int dimy = 32;
		dim3 block(dimx, dimy);
		dim3 grid((nd + block.x - 1) / block.x, (nd + block.y - 1) / block.y);

		double iStart = cpuSecond();
		calc_coeff_cbct << < grid, block >> > (d_res, nd, center, w, theta, sdd, rotcount);
		CHECK(hipDeviceSynchronize());
		double iElaps = cpuSecond() - iStart;

		CHECK(hipGetLastError());
		CHECK(hipMemcpy(result, d_res, nBytes, hipMemcpyDeviceToHost));

		CHECK(hipFree(d_res));

		//printf("Elapsed: %lf [s] \n", iElaps);

		hipDeviceReset();

		return;
	}

	int calc_sysmat2(float* elem, int* rowptr, int* colind, const int nv, const int nd, const int center, const int sdd) {

		float area = 0;
		float theta = 0;
		int rotatecount = 0;
		bool firstelem = true;

		int nonzero = 0;
		float* tmpmat = (float*)malloc(sizeof(float) * nd * nd);

		FILE* fp;

		fp = fopen("C:\\Users\\takum\\Dropbox\\Aoki_Lab\\util\\Reconstructor\\output\\sysmatgpu.csv", "w");


		int dev = 0;
		hipDeviceProp_t deviceprop;
		CHECK(hipGetDeviceProperties(&deviceprop, dev));
		CHECK(hipSetDevice(dev));

		int nxy = nd * nd;
		int nBytes = nxy * sizeof(float);

		float* d_res;
		CHECK(hipMalloc((void**)&d_res, nBytes));

		//CHECK(hipMemcpy(d_res, tmpmat, nBytes, hipMemcpyHostToDevice));

		int dimx = 32;
		int dimy = 32;
		dim3 block(dimx, dimy);
		dim3 grid((nd + block.x - 1) / block.x, (nd + block.y - 1) / block.y);

		char str[100000];
		char buf[24];

		str[0] = '\0';

		for (int v = 0; v < nv; v++)
		{

			printf("\r%d / %d", v, nv);

			if (theta >= PI / 4)
			{ //���e�p��45�x�𒴂�����摜��90�x�E��]�������e�p�� - 45�x��
				rotatecount++;
				theta -= PI / 2;
			}

			for (int w = 0; w < nd; w++)
			{

				double iStart = cpuSecond();
				calc_coeff_cbct << < grid, block >> > (d_res, nd, center, w, theta, sdd, rotatecount);
				CHECK(hipDeviceSynchronize());
				double iElaps = cpuSecond() - iStart;

				CHECK(hipGetLastError());
				CHECK(hipMemcpy(tmpmat, d_res, nBytes, hipMemcpyDeviceToHost));

				//printf("Elapsed: %lf [s] \n", iElaps);

				firstelem = true;

				//printf("hoge");
				str[0] = '\0';

				for (int y = 0; y < nd; y++)
				{
					for (int x = 0; x < nd; x++)
					{
						area = tmpmat[y * nd + x];
						if (area != 0) {
							//if (nonzero == MAXMATERIALS - 1) {
							//	exit(1);
							//}
							elem[nonzero] = area;
							colind[nonzero] = nd * y + x;
							if (firstelem) {
								rowptr[nd * v + w] = nonzero;
								firstelem = false;
							}
							nonzero++;
						}
						snprintf(buf, 24, "%f,", area);
						strcat(str, buf);
						//fprintf(fp, "%f", area);
					}
				}

				strcat(str, "\n");
				fprintf(fp, str);

			}

			theta += 2 * PI / nv;
		}

		CHECK(hipFree(d_res));
		hipDeviceReset();

		fclose(fp);

		return nonzero;
	}

	double cpuSecond() {

		SYSTEMTIME st;
		GetLocalTime(&st);

		return ((double)st.wSecond + (double)st.wMilliseconds * 1.e-3);

	}
}
