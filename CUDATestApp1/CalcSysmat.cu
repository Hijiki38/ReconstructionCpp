#include "hip/hip_runtime.h"
#include "CalcSysmat.cuh"


#define CHECK(call)\
{\
    const hipError_t error = call;\
    if (error != hipSuccess)\
        {\
            printf("Error: %s:%d",__FILE__,__LINE__);\
            printf("code:%d, reason: %s\n", error, hipGetErrorString(error));\
            exit(1);\
        }\
}

namespace Reconstruction {

	__global__ void calc_coeff_test(float* result, const int nd, const int center,
		const int w, const float theta, const float sdd, const int rotcount) {

		unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
		unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
		unsigned int idx = iy * nd + ix;

		if (ix < nd && iy < nd) {
			result[idx] = 0;
		}

	}


	__global__ void calc_coeff(float* result, const int nd, const int center,
		const int w, const float theta, const float sdd, const int rotcount) {

		unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
		unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
		unsigned int idx = iy * nd + ix;

		float relx = (float)ix;
		float rely = (float)iy;
		float tmp_x;
		float offset_detector, intercept;

		float la1, la2, lb1, lb2, sa, sb;
		float a = 0.5; // = pixsize / 2
		float tan_angle = tanf(theta);
		float cos_angle = cosf(theta);


		if (ix < nd && iy < nd) {
			for (int i = 0; i < rotcount; i++) {
				tmp_x = relx;
				relx = 2 * center - rely - 1;
				rely = tmp_x;
			}
			//float point::get_relative(float _num) { return _num - center + 0.5; }

			relx = relx - center + 0.5;
			rely = center - rely - 0.5;

			offset_detector = (nd - w - 1 - center + 0.5) / cosf(theta); //offset of the detector
			intercept = offset_detector - rely + relx * tan_angle;

			la1 = a - (-a * tan_angle + intercept + a / cos_angle);
			la2 = a - (a * tan_angle + intercept + a / cos_angle);
			lb1 = a + (-a * tan_angle + intercept - a / cos_angle);
			lb2 = a + (a * tan_angle + intercept - a / cos_angle);

			if (la1 < 0) {
				if (la2 < 0) {
					sa = 0;
				}
				else {
					sa = a * la2 * la2 / (-la1 + la2);
				}
			}
			else if (la1 < 2 * a) {
				if (la2 < 0) {
					sa = a * la1 * la1 / (la1 - la2);
				}
				else if (la2 < 2 * a) {
					sa = a * (la1 + la2);
				}
				else {
					sa = a * (la1 + la2) - (la2 - 2 * a) * (la2 - 2 * a) / (2 * (la2 - la1));
				}
			}
			else {
				if (la2 < 2 * a) {
					sa = a * (la1 + la2) - (la1 - 2 * a) * (la1 - 2 * a) / (2 * (la1 - la2));
				}
				else {
					sa = (2 * a) * (2 * a);
				}
			}

			if (lb1 < 0) {
				if (lb2 < 0) {
					sb = 0;
				}
				else {
					sb = a * lb2 * lb2 / (-lb1 + lb2);
				}
			}
			else if (lb1 < 2 * a) {
				if (lb2 < 0) {
					sb = a * lb1 * lb1 / (lb1 -lb2);
				}
				else if (lb2 < 2 * a) {
					sb = a * (lb1 + lb2);
				}
				else {
					sb = a * (lb1 + lb2) - (lb2 - 2 * a) * (lb2 - 2 * a) / (2 * (lb2 - lb1));
				}
			}
			else {
				if (lb2 < 2 * a) {
					sb = a * (lb1 + lb2) - (lb1 - 2 * a) * (lb1 - 2 * a) / (2 * (lb1 - lb2));
				}
				else {
					sb = (2 * a) * (2 * a);
				}
			}

			result[idx] = (2 * a) * (2 * a) - (sa + sb);
		}
	}


	__global__ void calc_coeff_cbct(float* result, const int nd, const int center,
		const int w, const float theta, const float sdd, const int rotcount) {

		unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
		unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
		unsigned int idx = iy * nd + ix;

		float relx = ix;
		float rely = iy;
		float tmp_x;
		float offset_detector, phi, intercept;

		float la1, la2, lb1, lb2, sa, sb;
		float a = 0.5; // = pixsize / 2
		float tan_angle;
		float tan_delta = a / sdd;

		if (ix < nd && iy < nd) {
			for (int i = 0; i < rotcount; i++) {
				tmp_x = relx;
				relx = 2 * center - rely - 1;
				rely = tmp_x;
			}
			//float point::get_relative(float _num) { return _num - center + 0.5; }

			relx = relx - center + 0.5;
			rely = center - rely - 0.5;

			offset_detector = (nd - w - 1 - center + 0.5) / cosf(theta); //offset of the detector
			phi = atan2f(offset_detector, sdd);

			tan_angle = tanf(theta + phi);
			intercept = offset_detector - rely + relx * tan_angle;

			la1 = a - (-a * (tan_angle + tan_delta) + intercept + a / sqrt(1 / (1 + (tan_angle + tan_delta) * (tan_angle + tan_delta))));
			la2 = a - (a * (tan_angle + tan_delta) + intercept + a / sqrt(1 / (1 + (tan_angle + tan_delta) * (tan_angle + tan_delta))));
			lb1 = a + (-a * (tan_angle - tan_delta) + intercept - a / sqrt(1 / (1 + (tan_angle - tan_delta) * (tan_angle - tan_delta))));
			lb2 = a + (a * (tan_angle - tan_delta) + intercept - a / sqrt(1 / (1 + (tan_angle - tan_delta) * (tan_angle - tan_delta))));

			if (la1 < 0) {
				if (la2 < 0) {
					sa = 0;
				}
				else {
					sa = a * la2 * la2 / (-la1 + la2);
				}
			}
			else if (la1 < 2 * a) {
				if (la2 < 0) {
					sa = a * la1 * la1 / (la1 - la2);
				}
				else if (la2 < 2 * a) {
					sa = a * (la1 + la2);
				}
				else {
					sa = a * (la1 + la2) - (la2 - 2 * a) * (la2 - 2 * a) / (2 * (la2 - la1));
				}
			}
			else {
				if (la2 < 2 * a) {
					sa = a * (la1 + la2) - (la1 - 2 * a) * (la1 - 2 * a) / (2 * (la1 - la2));
				}
				else {
					sa = (2 * a) * (2 * a);
				}
			}

			if (lb1 < 0) {
				if (lb2 < 0) {
					sb = 0;
				}
				else {
					sb = a * lb2 * lb2 / (-lb1 + lb2);
				}
			}
			else if (lb1 < 2 * a) {
				if (lb2 < 0) {
					sb = a * lb1 * lb1 / (lb1 - lb2);
				}
				else if (lb2 < 2 * a) {
					sb = a * (lb1 + lb2);
				}
				else {
					sb = a * (lb1 + lb2) - (lb2 - 2 * a) * (lb2 - 2 * a) / (2 * (lb2 - lb1));
				}
			}
			else {
				if (lb2 < 2 * a) {
					sb = a * (lb1 + lb2) - (lb1 - 2 * a) * (lb1 - 2 * a) / (2 * (lb1 - lb2));
				}
				else {
					sb = (2 * a) * (2 * a);
				}
			}

			result[idx] = (2 * a) * (2 * a) - (sa + sb);
		}
	}

	__global__ void calc_coeff_cbct_l(float* result, const int nd, const int center,
		const float theta, const float phi, const float sod_norm, const float sdd_norm, const int rotcount) {

		unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
		unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
		unsigned int idx = iy * nd + ix;

		float relx = ix;
		float rely = iy;
		float tmp_x;
		float offset_detector;
		float intercept;

		float l0, l1, k;
		float a = 0.5; //pixsize / 2
		float tan_angle;
		float tan_delta = a / sdd_norm;

		float pi = 3.141592;

		if (ix < nd && iy < nd) {
			for (int i = 0; i < rotcount; i++) {
				tmp_x = relx;
				relx = 2 * center - rely - 1;
				rely = tmp_x;
			}
			//float point::get_relative(float _num) { return _num - center + 0.5; }

			relx = relx - center + 0.5;
			rely = center - rely - 0.5;

			tan_angle = tanf(theta + phi);
			//intercept = offset_detector - rely + relx * tan_angle; //���Y�s�N�Z����ł̂��ؕ�
			intercept = sod_norm * tanf(phi) / (cosf(phi) * cosf(theta + phi)) - rely + relx * tan_angle; //���Y�s�N�Z����ł̂��ؕ�

			k = 0;

			if (theta + phi < pi / 2 && theta + phi >= 0) {  //0~45
				l0 = (-a - intercept) / tan_angle;
				l1 = (a - intercept) / tan_angle;
				if (l1 >= a && l0 < -a) {
					k = 1;
				}
				else if (l1 < a && l0 < -a) {
					k = 0.5 + l1 / (2 * a); //might be negative
				}
				else {
					k = 0.5 - l0 / (2 * a); //might be negative
				}
			}
			else if (theta + phi > -pi / 2 && theta + phi < 0) { //-45~0
				l0 = (a - intercept) / tan_angle;
				l1 = (-a - intercept) / tan_angle;
				if (l1 >= a && l0 < -a) {
					k = 1;
				}
				else if (l1 < a && l0 < -a) {
					k = 0.5 + l1 / (2 * a); //might be negative
				}
				else {
					k = 0.5 - l0 / (2 * a); //might be negative
				}
			}
			//else if (theta + phi > pi / 2) { //45~
			//	l0 = -a * tan_angle + intercept;
			//	l1 = a * tan_angle + intercept;
			//	if (l1 >= a && l0 < -a) {
			//		k = 1;
			//	}
			//	else if (l1 < a && l0 < -a) {
			//		k = 0.5 + l1 / (2 * a); //might be negative
			//	}
			//	else {
			//		k = 0.5 - l0 / (2 * a); //might be negative
			//	}
			//}
			//else { //~-45
			//	l0 = -a * tan_angle + intercept;
			//	l1 = a * tan_angle + intercept;
			//	if (l0 >= a && l1 < -a) {
			//		k = 1;
			//	}
			//	else if (l0 < a && l1 < -a) {
			//		k = 0.5 + l1 / (2 * a); //might be negative
			//	}
			//	else {
			//		k = 0.5 - l0 / (2 * a); //might be negative
			//	}
			//}

			result[idx] = k / cosf(theta + phi);
		}
	}

	void deviceinit() {

	}

	void devicereset() {
		CHECK(hipDeviceReset());
	}

	int calc_sysmat(float* elem, int* rowptr, int* colind, const int nv, const int nd, const int center, const float sdd, const bool write_sysmat) {

		float area = 0;
		float theta = 0;
		int rotatecount = 0;
		bool firstelem = true;

		int nonzero = 0;
		float* tmpmat = (float*)malloc(sizeof(float) * nd * nd);


		int dev = 0;
		hipDeviceProp_t deviceprop;
		CHECK(hipGetDeviceProperties(&deviceprop, dev));
		CHECK(hipSetDevice(dev));

		int nxy = nd * nd;
		int nBytes = nxy * sizeof(float);

		float* d_res;
		printf("\ncudamalloc");
		CHECK(hipMalloc((void**)&d_res, nBytes));
		printf("\ncompleted");
		//CHECK(hipMemcpy(d_res, tmpmat, nBytes, hipMemcpyHostToDevice));

		int dimx = 32;
		int dimy = 32;
		dim3 block(dimx, dimy);
		dim3 grid((nd + block.x - 1) / block.x, (nd + block.y - 1) / block.y);

		char str[100000];
		char buf[24];

		str[0] = '\0';

		for (int v = 0; v < nv; v++)
		{

			printf("\r%d / %d", v, nv);

			if (theta >= PI / 4)
			{ //���e�p��45�x�𒴂�����摜��90�x�E��]�������e�p�� - 45�x��
				rotatecount++;
				theta -= PI / 2;
			}

			for (int w = 0; w < nd; w++)
			{

				double iStart = cpuSecond();
				calc_coeff_cbct << < grid, block >> > (d_res, nd, center, w, theta, sdd, rotatecount);
				CHECK(hipDeviceSynchronize());
				double iElaps = cpuSecond() - iStart;

				CHECK(hipGetLastError());
				CHECK(hipMemcpy(tmpmat, d_res, nBytes, hipMemcpyDeviceToHost));

				//printf("Elapsed: %lf [s] \n", iElaps);

				firstelem = true;

				//printf("hoge");
				str[0] = '\0';

				for (int y = 0; y < nd; y++)
				{
					for (int x = 0; x < nd; x++)
					{
						area = tmpmat[y * nd + x];
						if (area != 0) {
							//if (nonzero == MAXMATERIALS - 1) {
							//	exit(1);
							//}
							elem[nonzero] = area;
							colind[nonzero] = nd * y + x;
							if (firstelem) {
								rowptr[nd * v + w] = nonzero;
								firstelem = false;
							}
							nonzero++;
						}

					}
				}

			}

			theta += 2 * PI / nv;
		}

		CHECK(hipFree(d_res));
		CHECK(hipDeviceReset());

		//printf("check elem[100]: %f\n", elem[100]);

		return nonzero;
	}

	int calc_sysmat_l(float* elem, int* rowptr, int* colind, const int nv, const int nd, const int center, const float sod, const float sdd, const bool write_sysmat) {

		float area = 0;
		float theta = 0;
		float thetatmp = 0;
		float phi = 0;
		int rotatecount = 0;
		int additionalrot = 0;
		bool firstelem = true;

		float offset_detector;

		int nonzero = 0;
		float* tmpmat = (float*)malloc(sizeof(float) * nd * nd);


		int dev = 0;
		hipDeviceProp_t deviceprop;
		CHECK(hipGetDeviceProperties(&deviceprop, dev));
		CHECK(hipSetDevice(dev));

		int nxy = nd * nd;
		int nBytes = nxy * sizeof(float);

		float* d_res;
		printf("\ncudamalloc");
		CHECK(hipMalloc((void**)&d_res, nBytes));
		printf("\ncompleted");
		//CHECK(hipMemcpy(d_res, tmpmat, nBytes, hipMemcpyHostToDevice));

		int dimx = 32;
		int dimy = 32;
		dim3 block(dimx, dimy);
		dim3 grid((nd + block.x - 1) / block.x, (nd + block.y - 1) / block.y);

		char str[100000];
		char buf[24];

		str[0] = '\0';

		for (int v = 0; v < nv; v++)
		{

			printf("\r%d / %d", v, nv);

			if (theta >= PI / 4)
			{ //���e�p��45�x�𒴂�����摜��90�x�E��]�������e�p�� - 45�x��
				rotatecount++;
				theta -= PI / 2;
			}


			for (int w = 0; w < nd; w++)
			{

				offset_detector = (nd - w - 1 - center + 0.5); //offset of the detector
				phi = atan2f(offset_detector, sdd);

				additionalrot = 0;
				thetatmp = theta;
				if (theta + phi >= PI / 4) {
					additionalrot = 1;
					thetatmp -= PI / 2;
				}
				else if (theta + phi < -PI / 4) {
					additionalrot = -1;
					thetatmp += PI / 2;
				}

				double iStart = cpuSecond();
				calc_coeff_cbct_l << < grid, block >> > (d_res, nd, center, thetatmp, phi, sod, sdd, rotatecount + additionalrot);
				CHECK(hipDeviceSynchronize());
				double iElaps = cpuSecond() - iStart;


				//	__global__ void calc_coeff_cbct_l(float* result, const int nd, const int center,
				//const float theta, const float phi, const float sod_norm, const float sdd_norm, const int rotcount);

				CHECK(hipGetLastError());
				CHECK(hipMemcpy(tmpmat, d_res, nBytes, hipMemcpyDeviceToHost));

				//printf("Elapsed: %lf [s] \n", iElaps);

				firstelem = true;

				//printf("hoge");
				str[0] = '\0';

				for (int y = 0; y < nd; y++)
				{
					for (int x = 0; x < nd; x++)
					{
						area = tmpmat[y * nd + x];
						if (area > 0) {
							//if (nonzero == MAXMATERIALS - 1) {
							//	exit(1);
							//}
							elem[nonzero] = area;
							colind[nonzero] = nd * y + x;
							if (firstelem) {
								rowptr[nd * v + w] = nonzero;
								firstelem = false;
							}
							nonzero++;
						}

					}
				}

			}

			theta += 2 * PI / nv;
		}

		CHECK(hipFree(d_res));
		CHECK(hipDeviceReset());

		//printf("check elem[100]: %f\n", elem[100]);

		return nonzero;
	}

	int calc_sysmat2(float* elem, int* rowptr, int* colind, const int v_start, const int v_size, const int nv, const int nd, const int center, const float sdd, const bool init, const bool write_sysmat) {

		float area = 0;
		float theta = 0;
		int rotatecount = 0;
		bool firstelem = true;

		int nonzero = 0;
		float* tmpmat = (float*)malloc(sizeof(float) * nd * nd);



		if (write_sysmat) {
			FILE* fp;
			fp = fopen("C:\\Users\\takum\\Dropbox\\Aoki_Lab\\util\\Reconstructor\\output\\sysmatgpu.csv", "w");
		}

		if (init) {
			int dev = 0;
			hipDeviceProp_t deviceprop;
			CHECK(hipGetDeviceProperties(&deviceprop, dev));
			CHECK(hipSetDevice(dev));
		}


		int nxy = nd * nd;
		int nBytes = nxy * sizeof(float);

		float* d_res;
		CHECK(hipMalloc((void**)&d_res, nBytes));

		int dimx = 32;
		int dimy = 32;
		dim3 block(dimx, dimy);
		dim3 grid((nd + block.x - 1) / block.x, (nd + block.y - 1) / block.y);

		//char str[100000];
		//char buf[24];

		//str[0] = '\0';



		for (int i = 0; i < v_start; i++) {
			if (theta >= PI / 4)
			{ //���e�p��45�x�𒴂�����摜��90�x�E��]�������e�p�� - 45�x��
				rotatecount++;
				theta -= PI / 2;
			}
			theta += 2 * PI / nv;
		}

		for (int v = v_start; v < v_start + v_size; v++)
		{

			printf("\r%d / %d to %d  (theta: %f)", v, v_start, v_start + v_size, theta);

			if (theta >= PI / 4)
			{ //���e�p��45�x�𒴂�����摜��90�x�E��]�������e�p�� - 45�x��
				rotatecount++;
				theta -= PI / 2;
			}

			for (int w = 0; w < nd; w++)
			{

				//double iStart = cpuSecond();
				//calc_coeff << < grid, block >> > (d_res, nd, center, w, theta, sdd, rotatecount);
				//printf("\ncalling kernel");
				calc_coeff << < grid, block >> > (d_res, nd, center, w, theta, sdd, rotatecount);
				//printf("... finished\n");
				CHECK(hipDeviceSynchronize());
				//printf("... finished\n");
				//double iElaps = cpuSecond() - iStart;

				CHECK(hipGetLastError());

				//printf("\n hipMemcpy: %d", w);
				CHECK(hipMemcpy(tmpmat, d_res, nBytes, hipMemcpyDeviceToHost));
				//printf("... finished\n");
				
				//printf("Elapsed: %lf [s] \n", iElaps);

				firstelem = true;

				//printf("hoge");
				//if (write_sysmat) {
				//	str[0] = '\0';
				//}

				//printf("\n convert to spmat");

				for (int y = 0; y < nd; y++)
				{
					for (int x = 0; x < nd; x++)
					{
						area = tmpmat[y * nd + x];
						//if (area != 0 && (x - center) * (x - center) + (y - center) * (y - center) < center * center) {
						if (area != 0) {
							elem[nonzero] = area;
							colind[nonzero] = nd * y + x;
							if (firstelem) {
								rowptr[nd * (v - v_start) + w] = nonzero;
								firstelem = false;
							}
							nonzero++;
						}
						//if (write_sysmat) {
						//	snprintf(buf, 24, "%f,", area);
						//	strcat(str, buf);
						//}
					}
				}

				//printf("... finished\n");

				//if (write_sysmat) {
				//	strcat(str, "\n");
				//	fprintf(fp, str);
				//}


			}

			theta += 2 * PI / nv;
		}

		printf("check elem[100]: %f\n", elem[100]);

		//printf("\ncudafree\n");
		CHECK(hipFree(d_res));
		//printf("... finished\n");
		//printf("\ncudadevicereset\n");
		CHECK(hipDeviceReset());
		//printf("... finished\n");

		//if (write_sysmat) {
		//	fclose(fp);
		//}

		// for debug
		//exit(0);



		//printf("\nfree tmpmat\n");
		free(tmpmat);
		//printf("... finished\n");

		////for debug
		//elem[0] = 0;
		//rowptr[0] = 0;
		//colind[0] = 0;
		//return 0;
		////


		//printf("\n %d !!!!!!!!!!!!!!!!!!!!!!!! \n", nonzero);

		return nonzero;
	}

	double cpuSecond() {

		SYSTEMTIME st;
		GetLocalTime(&st);

		return ((double)st.wSecond + (double)st.wMilliseconds * 1.e-3);

	}
}
