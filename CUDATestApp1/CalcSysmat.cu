#include "hip/hip_runtime.h"
#include "CalcSysmat.cuh"


#define CHECK(call)\
{\
    const hipError_t error = call;\
    if (error != hipSuccess)\
        {\
            printf("Error: %s:%d",__FILE__,__LINE__);\
            printf("code:%d, reason: %s\n", error, hipGetErrorString(error));\
            exit(1);\
        }\
}

namespace Reconstruction {

	__global__ void calc_coeff_test(float* result, const int nd, const int center,
		const int w, const float theta, const float sdd, const int rotcount) {

		unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
		unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
		unsigned int idx = iy * nd + ix;

		if (ix < nd && iy < nd) {
			result[idx] = 0;
		}

	}


	__global__ void calc_coeff(float* result, const int nd, const int center,
		const int w, const float theta, const float sdd, const int rotcount) {

		unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
		unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
		unsigned int idx = iy * nd + ix;

		float relx = (float)ix;
		float rely = (float)iy;
		float tmp_x;
		float offset_detector, intercept;

		float la1, la2, lb1, lb2, sa, sb;
		float a = 0.5; // = pixsize / 2
		float tan_angle = tanf(theta);
		float cos_angle = cosf(theta);


		if (ix < nd && iy < nd) {
			for (int i = 0; i < rotcount; i++) {
				tmp_x = relx;
				relx = 2 * center - rely - 1;
				rely = tmp_x;
			}
			//float point::get_relative(float _num) { return _num - center + 0.5; }

			relx = relx - center + 0.5;
			rely = center - rely - 0.5;

			offset_detector = (nd - w - 1 - center + 0.5) / cosf(theta); //offset of the detector
			intercept = offset_detector - rely + relx * tan_angle;

			la1 = a - (-a * tan_angle + intercept + a / cos_angle);
			la2 = a - (a * tan_angle + intercept + a / cos_angle);
			lb1 = a + (-a * tan_angle + intercept - a / cos_angle);
			lb2 = a + (a * tan_angle + intercept - a / cos_angle);

			if (la1 < 0) {
				if (la2 < 0) {
					sa = 0;
				}
				else {
					sa = a * la2 * la2 / (-la1 + la2);
				}
			}
			else if (la1 < 2 * a) {
				if (la2 < 0) {
					sa = a * la1 * la1 / (la1 - la2);
				}
				else if (la2 < 2 * a) {
					sa = a * (la1 + la2);
				}
				else {
					sa = a * (la1 + la2) - (la2 - 2 * a) * (la2 - 2 * a) / (2 * (la2 - la1));
				}
			}
			else {
				if (la2 < 2 * a) {
					sa = a * (la1 + la2) - (la1 - 2 * a) * (la1 - 2 * a) / (2 * (la1 - la2));
				}
				else {
					sa = (2 * a) * (2 * a);
				}
			}

			if (lb1 < 0) {
				if (lb2 < 0) {
					sb = 0;
				}
				else {
					sb = a * lb2 * lb2 / (-lb1 + lb2);
				}
			}
			else if (lb1 < 2 * a) {
				if (lb2 < 0) {
					sb = a * lb1 * lb1 / (lb1 -lb2);
				}
				else if (lb2 < 2 * a) {
					sb = a * (lb1 + lb2);
				}
				else {
					sb = a * (lb1 + lb2) - (lb2 - 2 * a) * (lb2 - 2 * a) / (2 * (lb2 - lb1));
				}
			}
			else {
				if (lb2 < 2 * a) {
					sb = a * (lb1 + lb2) - (lb1 - 2 * a) * (lb1 - 2 * a) / (2 * (lb1 - lb2));
				}
				else {
					sb = (2 * a) * (2 * a);
				}
			}

			result[idx] = (2 * a) * (2 * a) - (sa + sb);
		}
	}


	__global__ void calc_coeff_cbct(float* result, const int nd, const int center,
		const int w, const float theta, const float sdd, const int rotcount) {

		unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
		unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
		unsigned int idx = iy * nd + ix;

		float relx = ix;
		float rely = iy;
		float tmp_x;
		float offset_detector, phi, intercept;

		float la1, la2, lb1, lb2, sa, sb;
		float a = 0.5; // = pixsize / 2
		float tan_angle;
		float tan_delta = a / sdd;

		if (ix < nd && iy < nd) {
			for (int i = 0; i < rotcount; i++) {
				tmp_x = relx;
				relx = 2 * center - rely - 1;
				rely = tmp_x;
			}
			//float point::get_relative(float _num) { return _num - center + 0.5; }

			relx = relx - center + 0.5;
			rely = center - rely - 0.5;

			offset_detector = (nd - w - 1 - center + 0.5) / cosf(theta); //offset of the detector
			phi = atan2f(offset_detector, sdd);

			tan_angle = tanf(theta + phi);
			intercept = offset_detector - rely + relx * tan_angle;

			la1 = a - (-a * (tan_angle + tan_delta) + intercept + a / sqrt(1 / (1 + (tan_angle + tan_delta) * (tan_angle + tan_delta))));
			la2 = a - (a * (tan_angle + tan_delta) + intercept + a / sqrt(1 / (1 + (tan_angle + tan_delta) * (tan_angle + tan_delta))));
			lb1 = a + (-a * (tan_angle - tan_delta) + intercept - a / sqrt(1 / (1 + (tan_angle - tan_delta) * (tan_angle - tan_delta))));
			lb2 = a + (a * (tan_angle - tan_delta) + intercept - a / sqrt(1 / (1 + (tan_angle - tan_delta) * (tan_angle - tan_delta))));

			if (la1 < 0) {
				if (la2 < 0) {
					sa = 0;
				}
				else {
					sa = a * la2 * la2 / (-la1 + la2);
				}
			}
			else if (la1 < 2 * a) {
				if (la2 < 0) {
					sa = a * la1 * la1 / (la1 - la2);
				}
				else if (la2 < 2 * a) {
					sa = a * (la1 + la2);
				}
				else {
					sa = a * (la1 + la2) - (la2 - 2 * a) * (la2 - 2 * a) / (2 * (la2 - la1));
				}
			}
			else {
				if (la2 < 2 * a) {
					sa = a * (la1 + la2) - (la1 - 2 * a) * (la1 - 2 * a) / (2 * (la1 - la2));
				}
				else {
					sa = (2 * a) * (2 * a);
				}
			}

			if (lb1 < 0) {
				if (lb2 < 0) {
					sb = 0;
				}
				else {
					sb = a * lb2 * lb2 / (-lb1 + lb2);
				}
			}
			else if (lb1 < 2 * a) {
				if (lb2 < 0) {
					sb = a * lb1 * lb1 / (lb1 - lb2);
				}
				else if (lb2 < 2 * a) {
					sb = a * (lb1 + lb2);
				}
				else {
					sb = a * (lb1 + lb2) - (lb2 - 2 * a) * (lb2 - 2 * a) / (2 * (lb2 - lb1));
				}
			}
			else {
				if (lb2 < 2 * a) {
					sb = a * (lb1 + lb2) - (lb1 - 2 * a) * (lb1 - 2 * a) / (2 * (lb1 - lb2));
				}
				else {
					sb = (2 * a) * (2 * a);
				}
			}

			result[idx] = (2 * a) * (2 * a) - (sa + sb);
		}
	}

	void deviceinit() {

	}

	void devicereset() {
		CHECK(hipDeviceReset());
	}

	int calc_sysmat(float* elem, int* rowptr, int* colind, const int nv, const int nd, const int center, const float sdd, const bool write_sysmat) {

		float area = 0;
		float theta = 0;
		int rotatecount = 0;
		bool firstelem = true;

		int nonzero = 0;
		float* tmpmat = (float*)malloc(sizeof(float) * nd * nd);


		int dev = 0;
		hipDeviceProp_t deviceprop;
		CHECK(hipGetDeviceProperties(&deviceprop, dev));
		CHECK(hipSetDevice(dev));

		int nxy = nd * nd;
		int nBytes = nxy * sizeof(float);

		float* d_res;
		printf("\ncudamalloc");
		CHECK(hipMalloc((void**)&d_res, nBytes));
		printf("\ncompleted");
		//CHECK(hipMemcpy(d_res, tmpmat, nBytes, hipMemcpyHostToDevice));

		int dimx = 32;
		int dimy = 32;
		dim3 block(dimx, dimy);
		dim3 grid((nd + block.x - 1) / block.x, (nd + block.y - 1) / block.y);

		char str[100000];
		char buf[24];

		str[0] = '\0';

		for (int v = 0; v < nv; v++)
		{

			printf("\r%d / %d", v, nv);

			if (theta >= PI / 4)
			{ //���e�p��45�x�𒴂�����摜��90�x�E��]�������e�p�� - 45�x��
				rotatecount++;
				theta -= PI / 2;
			}

			for (int w = 0; w < nd; w++)
			{

				double iStart = cpuSecond();
				calc_coeff_cbct << < grid, block >> > (d_res, nd, center, w, theta, sdd, rotatecount);
				CHECK(hipDeviceSynchronize());
				double iElaps = cpuSecond() - iStart;

				CHECK(hipGetLastError());
				CHECK(hipMemcpy(tmpmat, d_res, nBytes, hipMemcpyDeviceToHost));

				//printf("Elapsed: %lf [s] \n", iElaps);

				firstelem = true;

				//printf("hoge");
				str[0] = '\0';

				for (int y = 0; y < nd; y++)
				{
					for (int x = 0; x < nd; x++)
					{
						area = tmpmat[y * nd + x];
						if (area != 0) {
							//if (nonzero == MAXMATERIALS - 1) {
							//	exit(1);
							//}
							elem[nonzero] = area;
							colind[nonzero] = nd * y + x;
							if (firstelem) {
								rowptr[nd * v + w] = nonzero;
								firstelem = false;
							}
							nonzero++;
						}

					}
				}

			}

			theta += 2 * PI / nv;
		}

		CHECK(hipFree(d_res));
		CHECK(hipDeviceReset());

		//printf("check elem[100]: %f\n", elem[100]);

		return nonzero;
	}

	int calc_sysmat2(float* elem, int* rowptr, int* colind, const int v_start, const int v_size, const int nv, const int nd, const int center, const float sdd, const bool init, const bool write_sysmat) {

		float area = 0;
		float theta = 0;
		int rotatecount = 0;
		bool firstelem = true;

		int nonzero = 0;
		float* tmpmat = (float*)malloc(sizeof(float) * nd * nd);



		if (write_sysmat) {
			FILE* fp;
			fp = fopen("C:\\Users\\takum\\Dropbox\\Aoki_Lab\\util\\Reconstructor\\output\\sysmatgpu.csv", "w");
		}

		if (init) {
			int dev = 0;
			hipDeviceProp_t deviceprop;
			CHECK(hipGetDeviceProperties(&deviceprop, dev));
			CHECK(hipSetDevice(dev));
		}


		int nxy = nd * nd;
		int nBytes = nxy * sizeof(float);

		float* d_res;
		CHECK(hipMalloc((void**)&d_res, nBytes));

		int dimx = 32;
		int dimy = 32;
		dim3 block(dimx, dimy);
		dim3 grid((nd + block.x - 1) / block.x, (nd + block.y - 1) / block.y);

		//char str[100000];
		//char buf[24];

		//str[0] = '\0';



		for (int i = 0; i < v_start; i++) {
			if (theta >= PI / 4)
			{ //���e�p��45�x�𒴂�����摜��90�x�E��]�������e�p�� - 45�x��
				rotatecount++;
				theta -= PI / 2;
			}
			theta += 2 * PI / nv;
		}

		for (int v = v_start; v < v_start + v_size; v++)
		{

			printf("\r%d / %d to %d  (theta: %f)", v, v_start, v_start + v_size, theta);

			if (theta >= PI / 4)
			{ //���e�p��45�x�𒴂�����摜��90�x�E��]�������e�p�� - 45�x��
				rotatecount++;
				theta -= PI / 2;
			}

			for (int w = 0; w < nd; w++)
			{

				//double iStart = cpuSecond();
				//calc_coeff << < grid, block >> > (d_res, nd, center, w, theta, sdd, rotatecount);
				//printf("\ncalling kernel");
				calc_coeff << < grid, block >> > (d_res, nd, center, w, theta, sdd, rotatecount);
				//printf("... finished\n");
				CHECK(hipDeviceSynchronize());
				//printf("... finished\n");
				//double iElaps = cpuSecond() - iStart;

				CHECK(hipGetLastError());

				//printf("\n hipMemcpy: %d", w);
				CHECK(hipMemcpy(tmpmat, d_res, nBytes, hipMemcpyDeviceToHost));
				//printf("... finished\n");
				
				//printf("Elapsed: %lf [s] \n", iElaps);

				firstelem = true;

				//printf("hoge");
				//if (write_sysmat) {
				//	str[0] = '\0';
				//}

				//printf("\n convert to spmat");

				for (int y = 0; y < nd; y++)
				{
					for (int x = 0; x < nd; x++)
					{
						area = tmpmat[y * nd + x];
						if (area != 0 && (x - center) * (x - center) + (y - center) * (y - center) > center * center ) {
							elem[nonzero] = area;
							colind[nonzero] = nd * y + x;
							if (firstelem) {
								rowptr[nd * (v - v_start) + w] = nonzero;
								firstelem = false;
							}
							nonzero++;
						}
						//if (write_sysmat) {
						//	snprintf(buf, 24, "%f,", area);
						//	strcat(str, buf);
						//}
					}
				}

				//printf("... finished\n");

				//if (write_sysmat) {
				//	strcat(str, "\n");
				//	fprintf(fp, str);
				//}


			}

			theta += 2 * PI / nv;
		}

		printf("check elem[100]: %f\n", elem[100]);

		//printf("\ncudafree\n");
		CHECK(hipFree(d_res));
		//printf("... finished\n");
		//printf("\ncudadevicereset\n");
		CHECK(hipDeviceReset());
		//printf("... finished\n");

		//if (write_sysmat) {
		//	fclose(fp);
		//}

		// for debug
		//exit(0);



		//printf("\nfree tmpmat\n");
		free(tmpmat);
		//printf("... finished\n");

		////for debug
		//elem[0] = 0;
		//rowptr[0] = 0;
		//colind[0] = 0;
		//return 0;
		////


		//printf("\n %d !!!!!!!!!!!!!!!!!!!!!!!! \n", nonzero);

		return nonzero;
	}

	double cpuSecond() {

		SYSTEMTIME st;
		GetLocalTime(&st);

		return ((double)st.wSecond + (double)st.wMilliseconds * 1.e-3);

	}
}
