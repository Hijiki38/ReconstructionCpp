#include "hip/hip_runtime.h"
#include "CalcMat.cuh"


#define CHECK(call)\
{\
    const hipError_t error = call;\
    if (error != hipSuccess)\
        {\
            printf("Error: %s:%d",__FILE__,__LINE__);\
            printf("code:%d, reason: %s\n", error, hipGetErrorString(error));\
            exit(1);\
        }\
}

namespace Reconstruction {

	__global__ void product(float* a, float* b, float* res, const int nxa, const int nya, const int nxb) {

		unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
		unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
		unsigned int idx = iy * nxb + ix;

		if (ix < nxb && iy < nya) {
			res[idx] = 0;
			for (int i = 0; i < nxa; i++) {
				res[idx] += a[iy * nxa + i] * b[i * nxb + ix];
			}
		}
	}

	__global__ void init(float* a, const int nx, const int ny, const float value) {

		unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
		unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
		unsigned int idx = iy * nx + ix;

		if (ix < nx && iy < ny) {
			a[idx] = value;
		}
	}

	__global__ void transpose_col(float* a, float* res, const int nx, const int ny) {

		unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
		unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

		if (ix < nx && iy < ny) {
			res[iy * nx + ix] = a[ix * ny + iy];
		}
	}

	__global__ void intg_source(float* source, float* res, const int ni, const int nb, const int ne){ //n[i][b] += source[v_begin + i][b][e];

		unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
		unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
		unsigned int idx = iy * nb + ix;

		if (ix < nb && iy < ni) {
			res[idx] = 0;
			for (int i = 0; i < ne; i++) {
				res[idx] += source[(iy * nb * ne) + (ix * ne) + i];
			}
		}
	}

	__global__ void gradq(float* sysmat, float* n, float* nbar, float* grad_n, float* res, const int nd, const int ni, const int nb, const int nm) {

		unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
		unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
		unsigned int idx = iy * nm + ix; // nd*nd X nm

		if (ix < nm && iy < nd * nd) {
			res[idx] = 0;
			for (int i = 0; i < ni; i++) {
				for (int b = 0; b < nb; b++) {
					//res[idx] += -sysmat[i * nd * nd + iy] * (1 - (n[i * nb + b] / nbar[i * nb + b])) * grad_n[b * nm + ix];
					// 
					res[idx] += -sysmat[i * nd * nd + iy] * (1 - (n[i * nb + b] / nbar[i * nb + b])) * grad_n[(i * nb * nm) + (b * nm) + ix];
					//res[idx] += (1 - (n[i * nb + b] / nbar[i * nb + b])) * grad_n[(i * nb * nm) + (b * nm) + ix];
				}
			}
		}

	}

	__global__ void gradq2_1(float* source, float* matatn, float* lintg, float* res, const int ni, const int nb, const int ne, const int nm) {
		unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
		unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
		unsigned int idx = iy * nm + ix; // ni*nb X nm

		if (ix < nm && iy < ni * nb) {
			res[idx] = 0;
			for (int e = 0; e < ne; e++) {
				res[idx] += source[(iy / nb) * nb * ne + (iy % nb) * ne + e] * lintg[(iy / nb) * ne + e] * matatn[ix * ne + e];
			}
			//res[idx] /= 10;

		}
	}

	__global__ void gradq2_2(float* sysmat, float* n, float* nbar, float* suma, float* tmp, float* res, const int nd, const int ni, const int nb, const int nm) {
		unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
		unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
		unsigned int idx = iy * nm + ix; // nd*nd X nm

		if (ix < nm && iy < nd * nd) {
			res[idx] = 0;

			for (int i = 0; i < ni; i++) {
				for (int b = 0; b < nb; b++) {
					res[idx] += sysmat[i * nd * nd + iy] * ((n[i * nb + b] / nbar[i * nb + b]) - 1) * tmp[i * nb * nm + b * nm + ix];  // i b m
					//res[idx] += sysmat[i * nd * nd + iy] * ((n[i * nb + b] * tmp[i * nb * nm + b * nm + ix] / nbar[i * nb + b]) - tmp[i * nb * nm + b * nm + ix]);  // i b m
					//res[idx] += suma[i] * ((n[i * nb + b] * tmp[i * nb * nm + b * nm + ix] / nbar[i * nb + b]) - tmp[i * nb * nm + b * nm + ix]);  // i b m
					//res[idx] += (1 - (n[i * nb + b] / nbar[i * nb + b])) * grad_n[(i * nb * nm) + (b * nm) + ix];
				}
			}
		}
	}


	__global__ void nbard_k(float* source, float* lintg, float* res, float* bnmean, const int ni, const int nb, const int ne, const int nm) {
		unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
		unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
		unsigned int idx = iy * nb + ix; // ni X nb
		float tmp;

		if (ix < nb && iy < ni) {
			res[idx] = 0;
			for (int e = 0; e < ne; e++) {
				tmp = ((e * 0.1 + 15) - bnmean[iy % ni]);
				res[idx] += -source[iy * nb * ne + ix * ne + e] * tmp * tmp * lintg[(iy / nb) * ne + e] * 0.5;
			}

		}
	}

	__global__ void gradp(float* n, float* nbar, float* nbard, float* res, const int nd, const int ni, const int nb) {
		unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
		unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
		unsigned int idx = iy * nd + ix; // nd X nd

		if (ix < nd && iy < nd) {
			res[idx] = 0;

			for (int i = 0; i < ni; i++) {
				for (int b = 0; b < nb; b++) {
					res[idx] += (1 - (n[i * nb + b] / nbar[i * nb + b])) * -nbard[i * nb + b];  // i b m
				}
			}
		}
	}


	//for (int j = 0; j < nd * nd; j++) {
	//	//std::cout << "\r" << j << "/" << nd * nd;
	//	for (int k = 0; k < nm; k++) {
	//		for (int i = 0; i < block_size; i++) {
	//			for (int b = 0; b < nb; b++) {
	//				grad_q[j][k] += -smr[i][j] * (1 - (n[i][b] / nbar[i][b])) * grad_n[b][k];
	//			}
	//		}
	//	}
	//}

	__global__ void lintg(float* sysmat, float* matfrac, float* matatn, float* res, int nd, int ni, int ne, int nm) {

		unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
		unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
		unsigned int idx = iy * ne + ix;

		if (ix < ne && iy < ni) {
			res[idx] = 0;

			for (int j = 0; j < nd * nd; j++) {
				for (int k = 0; k < nm; k++) {
					res[idx] += sysmat[iy * nd * nd + j] * matfrac[j * nm + k] * matatn[k * ne + ix];
				}
			}
		}
		//lintg[i][e] += smr[i][j] * matfrac[j][k] * matatn[k][e];
	}

	//h[j][k][m] += smr[i][j] * (n[i][b] / nbar[i][b]) * suma[i] * source[v_begin + i][b][e] * matatn[k][e] * matatn[m][e] * lintg[i][e];

//res1[i][b][k][m]

	__global__ void hesseq1(float* source, float* matatn, float* lintg, float* res, const int ni, const int nb, const int ne, const int nm) {
		unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
		unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
		unsigned int idx = iy * nm * nm + ix; // ni*nb X nm*nm

		if (ix < nm * nm && iy < ni * nb) {
			res[idx] = 0;
			for (int e = 0; e < ne; e++) {
				//res[idx] += source[(iy / ni) * nb * ne + (iy % nb) * ne + e] * matatn[(ix % nm) * ne + e] * matatn[(ix / nm) * ne + e] * lintg[(iy / ni) * ne + e];
				res[idx] += source[(iy / nb) * nb * ne + (iy % nb) * ne + e] * matatn[(ix / nm) * ne + e] * matatn[(ix % nm) * ne + e] * lintg[(iy / nb) * ne + e];
			}
			//res[idx] /= 10;
		}
	}
	__global__ void hesseq2(float* sysmat, float* n, float* nbar, float* suma, float* tmp, float* res, const int nd, const int ni, const int nb, const int nm) {
		unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
		unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
		unsigned int idx = iy * nm * nm + ix; // nd*nd X nm*nm

		if (ix < nm * nm && iy < nd * nd) {
			res[idx] = 0;
			for (int i = 0; i < ni; i++) {
				for (int b = 0; b < nb; b++) {
					//res[idx] += sysmat[i * nd * nd + iy] * (n[i * nb + b] / nbar[i * nb + b]) * suma[i] * tmp[i * nb * nm * nm + b * nm * nm + (ix / nm) * nm + (ix % nm)];
					
					//res[idx] += sysmat[i * nd * nd + iy] * (n[i * nb + b] / nbar[i * nb + b]) * suma[i] * tmp[i * nb * nm * nm + b * nm * nm + ix];
					res[idx] += sysmat[i * nd * nd + iy] * suma[i] * tmp[i * nb * nm * nm + b * nm * nm + ix] * 0.5;
				}
			}
		}
	}

	__global__ void hesseqp(float* source, float* matatn, float* lintg, float* res, const int ni, const int nb, const int ne, const int nm) {
		unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
		unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
		unsigned int idx = iy * nm + ix; // ni*nb X nm

		if (ix < nm && iy < ni * nb) {
			res[idx] = 0;
			for (int e = 0; e < ne; e++) {
				//res[idx] += source[(iy / ni) * nb * ne + (iy % nb) * ne + e] * matatn[(ix % nm) * ne + e] * matatn[(ix / nm) * ne + e] * lintg[(iy / ni) * ne + e];
				res[idx] += source[(iy / nb) * nb * ne + (iy % nb) * ne + e] * matatn[ix * ne + e] * lintg[(iy / nb) * ne + e];
			}
		}
	}
	//__global__ void hesseqp2(float* nbar, float* nbard, float* suma, float* tmp, float* res, const int nd, const int ni, const int nb, const int nm) {
	//	unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
	//	unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
	//	unsigned int idx = iy * nm + ix; // nd*nd X nm

	//	if (ix < nm && iy < nd * nd) {
	//		res[idx] = 0;
	//		for (int i = 0; i < ni; i++) {
	//			for (int b = 0; b < nb; b++) {
	//				res[idx] += suma[i] * (nbard[i * nb + b] / nbar[i * nb + b]) * tmp[i * nb * nm + b * nm + ix] * 0.5;
	//			}
	//		}
	//	}
	//}

	//__global__ void hessep(float* n, float* nbar, float* nbard, float res, const int nd, const int ni, const int nb, const int nm) {
	//	unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
	//	unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
	//	unsigned int idx = iy * nm + ix; // nd*nd X nm

	//	if (ix < nm && iy < nd * nd) {
	//		res[idx] = 0;
	//		float tmp;
	//		for (int i = 0; i < ni; i++) {
	//			for (int b = 0; b < nb; b++) {
	//				tmp = nbard[i * nb + b] / nbar[i * nb + b];
	//				res[idx] += n[i * nb + b] * tmp * tmp + (1 - n[i * nb + b] / nbar[i * nb + b]) * nbard[i * nb + b] * 0.5;
	//			}
	//		}
	//	}
	//}

	//hesseqp1 << <grid1, block >> > (d_source, d_matatn, d_lintg, d_res1, ni, nb, ne, nm); // res1: ni*nb X nm
	//CHECK(hipDeviceSynchronize());
	//CHECK(hipGetLastError());

	//int count = 0;

	//hesseqp2 << <grid2, block >> > (d_nbar, d_suma, d_res1, d_res2, nd, ni, nb, nm); // res2: nd*nd X nm
	//CHECK(hipDeviceSynchronize());
	//CHECK(hipGetLastError());
	//CHECK(hipMemcpy(res2, d_res2, nBytes_res2, hipMemcpyDeviceToHost));


	//	hesseq2 << <grid2, block >> > (d_sysmat, d_n, d_nbar, d_suma, d_res1, d_res2, nd, ni, nb, nm);

	//for (int j = 0; j < nd * nd; j++) {
	//	std::cout << "\r" << j << "/" << nd * nd;
	//	for (int k = 0; k < nm; k++) {
	//		for (int m = 0; m < nm; m++) {
	//			for (int i = 0; i < block_size; i++) {
	//				for (int b = 0; b < nb; b++) {
	//					for (int e = 0; e < ne; e++) {
	//						h[j][k][m] += smr[i][j] * (n[i][b] / nbar[i][b]) * suma[i] * source[v_begin + i][b][e] * matatn[k][e] * matatn[m][e] * lintg[i][e];
	//					}
	//				}
	//			}
	//		}
	//	}
	//}



	void calc_product(float* a, float* b, float* res, int nxa, int nya, int nxb) {

		int dev = 0;
		hipDeviceProp_t deviceprop;
		CHECK(hipGetDeviceProperties(&deviceprop, dev));
		CHECK(hipSetDevice(dev));

		int nBytes_a = nya * nxa * sizeof(float);
		int nBytes_b = nxa * nxb * sizeof(float);
		int nBytes_r = nya * nxb * sizeof(float);

		float *d_a, *d_b, *d_res;
		CHECK(hipMalloc((void**)&d_a, nBytes_a));
		CHECK(hipMalloc((void**)&d_b, nBytes_b));
		CHECK(hipMalloc((void**)&d_res, nBytes_r));
		CHECK(hipMemcpy(d_a, a, nBytes_a, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_b, b, nBytes_b, hipMemcpyHostToDevice));
		//CHECK(hipMemcpy(d_res, res, nBytes_r, hipMemcpyHostToDevice));

		int dimx = 32;
		int dimy = 32;
		dim3 block(dimx, dimy);
		dim3 grid((nxb + block.x - 1) / block.x, (nya + block.y - 1) / block.y);


		//double iStart = cpuSecond();
		product <<< grid, block >>> (d_a, d_b, d_res, nxa, nya, nxb);
		CHECK(hipDeviceSynchronize());
		//double iElaps = cpuSecond() - iStart;

		CHECK(hipGetLastError());
		CHECK(hipMemcpy(res, d_res, nBytes_r, hipMemcpyDeviceToHost));

		CHECK(hipFree(d_a));
		CHECK(hipFree(d_b));
		CHECK(hipFree(d_res));
		CHECK(hipDeviceReset());
	}

	float** calc_product(float** a, float** b, int nxa, int nya, int nxb) {
		
		float* a1 = (float*)malloc(sizeof(float) * nya * nxa);
		float* b1 = (float*)malloc(sizeof(float) * nxa * nxb);
		float* r1 = (float*)malloc(sizeof(float) * nya * nxb);

		float** res = (float**)malloc(sizeof(float*) * nya);
		for (int i = 0; i < nya; i++) {
			res[i] = (float*)malloc(sizeof(float) * nxb);
		}

		for (int i = 0; i < nya; i++) {
			for (int j = 0; j < nxa; j++) {
				a1[i * nxa + j] = a[i][j];
			}
		}

		for (int i = 0; i < nxa; i++) {
			for (int j = 0; j < nxb; j++) {
				b1[i * nxb + j] = b[i][j];
			}
		}

		for (int i = 0; i < nya; i++) {
			for (int j = 0; j < nxb; j++) {
				r1[i * nxb + j] = res[i][j];
			}
		}

		calc_product(a1, b1, r1, nxa, nya, nxb);

		for (int i = 0; i < nya; i++) {
			for (int j = 0; j < nxb; j++) {
				res[i][j] = r1[i * nxb + j];
			}
		}

		free(a1);
		free(b1);
		free(r1);

		return res;
	}

	void init_matrix(float* a, int nx, int ny, float value) {

		int dev = 0;
		hipDeviceProp_t deviceprop;
		CHECK(hipGetDeviceProperties(&deviceprop, dev));
		CHECK(hipSetDevice(dev));

		int nBytes = nx * ny * sizeof(float);

		float* d_a;
		CHECK(hipMalloc((void**)&d_a, nBytes));
		CHECK(hipMemcpy(d_a, a, nBytes, hipMemcpyHostToDevice));

		int dimx = 32;
		int dimy = 32;
		dim3 block(dimx, dimy);
		dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);


		//double iStart = cpuSecond();
		init << < grid, block >> > (d_a, nx, ny, value);
		CHECK(hipDeviceSynchronize());
		//double iElaps = cpuSecond() - iStart;

		CHECK(hipGetLastError());
		CHECK(hipMemcpy(a, d_a, nBytes, hipMemcpyDeviceToHost));

		CHECK(hipFree(d_a));
		CHECK(hipDeviceReset());

	}

	void init_matrix(float** a, int nx, int ny, float value) {

		float* a1 = (float*)malloc(sizeof(float) * nx * ny);

		for (int i = 0; i < ny; i++) {
			for (int j = 0; j < nx; j++) {
				a1[i * nx + j] = a[i][j];
			}
		}

		init_matrix(a1, nx, ny, value);

		for (int i = 0; i < ny; i++) {
			for (int j = 0; j < nx; j++) {
				a[i][j] = a1[i * nx + j];
			}
		}

		free(a1);
	}

	float** calc_T(float** a, int nx, int ny) {

		float* a1 = (float*)malloc(sizeof(float) * nx * ny);

		float** res = (float**)malloc(sizeof(float*) * nx);
		for (int i = 0; i < nx; i++) {
			res[i] = (float*)malloc(sizeof(float) * ny);
		}

		for (int i = 0; i < ny; i++) {
			for (int j = 0; j < nx; j++) {
				a1[i * nx + j] = a[i][j];
			}
		}

		int dev = 0;
		hipDeviceProp_t deviceprop;
		CHECK(hipGetDeviceProperties(&deviceprop, dev));
		CHECK(hipSetDevice(dev));

		int nBytes = nx * ny * sizeof(float);

		float* d_a;
		float* d_res;
		CHECK(hipMalloc((void**)&d_a, nBytes));
		CHECK(hipMalloc((void**)&d_res, nBytes));
		CHECK(hipMemcpy(d_a, a, nBytes, hipMemcpyHostToDevice));

		int dimx = 32;
		int dimy = 32;
		dim3 block(dimx, dimy);
		dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);


		//double iStart = cpuSecond();
		transpose_col << < grid, block >> > (d_a, d_res, nx, ny);
		CHECK(hipDeviceSynchronize());
		//double iElaps = cpuSecond() - iStart;

		CHECK(hipGetLastError());
		CHECK(hipMemcpy(res, d_res, nBytes, hipMemcpyDeviceToHost));

		CHECK(hipFree(d_a));
		CHECK(hipFree(d_res));
		CHECK(hipDeviceReset());

		for (int i = 0; i < ny; i++) {
			for (int j = 0; j < nx; j++) {
				a[i][j] = a1[i * nx + j];
			}
		}

		free(a1);

		return a;

	}

	float** calc_n(float*** source, int ni, int nb, int ne, int v_begin) {  //n[i][b] += source[v_begin + i][b][e];

		float* source_block = (float*)malloc(sizeof(float) * ni * nb * ne);
		float* res1 = (float*)malloc(sizeof(float) * ni * nb);
		float** res = (float**)malloc(sizeof(float*) * ni);
		for (int i = 0; i < ni; i++) {
			res[i] = (float*)malloc(sizeof(float) * nb);
		}


		for (int i = 0; i < ni; i++) {
			for (int j = 0; j < nb; j++) {
				for (int k = 0; k < ne; k++) {
					source_block[(i * nb * ne) + (j * ne) + k] = source[i + v_begin][j][k];
				}
			}
		}

		int dev = 0;
		hipDeviceProp_t deviceprop;
		CHECK(hipGetDeviceProperties(&deviceprop, dev));
		CHECK(hipSetDevice(dev));

		int nBytes_s = ni * nb * ne * sizeof(float);
		int nBytes_r = ni * nb * sizeof(float);

		float* d_source;
		float* d_res;
		CHECK(hipMalloc((void**)&d_source, nBytes_s));
		CHECK(hipMalloc((void**)&d_res, nBytes_r));
		CHECK(hipMemcpy(d_source, source_block, nBytes_s, hipMemcpyHostToDevice));

		int dimx = 32;
		int dimy = 32;
		dim3 block(dimx, dimy);
		dim3 grid((nb + block.x - 1) / block.x, (ni + block.y - 1) / block.y);


		//double iStart = cpuSecond();
		intg_source << < grid, block >> > (d_source, d_res, ni, nb, ne);  //(float* source, float* res, int ni, int nb, int ne){
		CHECK(hipDeviceSynchronize());
		//double iElaps = cpuSecond() - iStart;

		CHECK(hipGetLastError());
		CHECK(hipMemcpy(res1, d_res, nBytes_r, hipMemcpyDeviceToHost));

		CHECK(hipFree(d_source));
		CHECK(hipFree(d_res));
		CHECK(hipDeviceReset());

		for (int i = 0; i < ni; i++) {
			for (int j = 0; j < nb; j++) {
				res[i][j] = res1[i * nb + j];
			}
		}

		free(source_block);
		free(res1);

		return res;
	
	}

	void calc_nbard(float** res, float*** source, float** lintg, float* bnmean, int nd, int ni, int nb, int ne, int nm, int v_begin) {

		int nBytes_source = sizeof(float) * ni * nb * ne;
		int nBytes_lintg = sizeof(float) * ni * ne;
		int nBytes_bnmean = sizeof(float) * nb;
		int nBytes_res1 = sizeof(float) * ni * nb; // ni*nb

		float* _source = (float*)malloc(nBytes_source);
		float* _lintg = (float*)malloc(nBytes_lintg);
		float* _bnmean = (float*)malloc(nBytes_bnmean);
		float* res1 = (float*)malloc(nBytes_res1);

		for (int i = 0; i < ni; i++) {
			for (int j = 0; j < nb; j++) {
				for (int k = 0; k < ne; k++) {
					_source[(i * nb * ne) + (j * ne) + k] = source[i + v_begin][j][k];
				}
			}
			for (int j = 0; j < ne; j++) {
				_lintg[(i * ne) + j] = lintg[i][j];
			}
		}

		for (int i = 0; i < nb; i++) {
			_bnmean[i] = bnmean[i];
		}

		int dev = 0;
		hipDeviceProp_t deviceprop;
		CHECK(hipGetDeviceProperties(&deviceprop, dev));
		CHECK(hipSetDevice(dev));

		float* d_source;
		float* d_lintg;
		float* d_bnmean;
		float* d_res1;

		CHECK(hipMalloc((void**)&d_source, nBytes_source));
		CHECK(hipMalloc((void**)&d_lintg, nBytes_lintg));
		CHECK(hipMalloc((void**)&d_bnmean, nBytes_bnmean));
		CHECK(hipMalloc((void**)&d_res1, nBytes_res1));

		CHECK(hipMemcpy(d_source, _source, nBytes_source, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_lintg, _lintg, nBytes_lintg, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_bnmean, bnmean, nBytes_bnmean, hipMemcpyHostToDevice));

		int dimx = 32;
		int dimy = 32;
		dim3 block(dimx, dimy);
		dim3 grid((nb + block.x - 1) / block.x, (ni + block.y - 1) / block.y);

		nbard_k << <grid, block >> > (d_source, d_lintg, d_res1, d_bnmean, ni, nb, ne, nm);
		CHECK(hipDeviceSynchronize());
		CHECK(hipGetLastError());
		CHECK(hipMemcpy(res1, d_res1, nBytes_res1, hipMemcpyDeviceToHost));


		CHECK(hipFree(d_source));
		CHECK(hipFree(d_lintg));
		CHECK(hipFree(d_bnmean));
		CHECK(hipFree(d_res1));
		CHECK(hipDeviceReset());

		for (int i = 0; i < ni; i++) {
			for (int j = 0; j < nb; j++) {
				res[i][j] += res1[i * nb + j];
			}
		}

		free(_source);
		free(_lintg);
		free(_bnmean);
		free(res1);

	}

	void calc_lintg(float** res, float** sysmat, float** matfrac, float** matatn, int nd, int ni, int ne, int nm, float pixsize) {

		int nBytes_sysmat = sizeof(float) * ni * nd * nd;
		int nBytes_matfrac = sizeof(float) * nd * nd * nm;
		int nBytes_matatn = sizeof(float) * nm * ne;
		int nBytes_res1 = sizeof(float) * nd * nd * ne;
		int nBytes_res2 = sizeof(float) * ni * ne;
		int nBytes_res = sizeof(float) * ni * ne;

		float* _sysmat = (float*)malloc(nBytes_sysmat);
		float* _matfrac = (float*)malloc(nBytes_matfrac);
		float* _matatn = (float*)malloc(nBytes_matatn);
		float* res2 = (float*)malloc(nBytes_res2);

		float* res_tmp = (float*)malloc(nBytes_res1);

		for (int i = 0; i < ni; i++) {
			for (int j = 0; j < nd * nd; j++) {
				_sysmat[(i * nd * nd) + j] = sysmat[i][j] * pixsize;
			}
		}
		for (int i = 0; i < nm; i++) {
			for (int j = 0; j < ne; j++) {
				_matatn[(i * ne) + j] = matatn[i][j];
			}
		}
		for (int i = 0; i < nd * nd; i++) {
			for (int j = 0; j < nm; j++) {
				_matfrac[(i * nm) + j] = matfrac[i][j];
			}
		}

		//float** res = (float**)malloc(sizeof(float*) * ni); //ni * ne
		//for (int i = 0; i < ni; i++) {
		//	res[i] = (float*)malloc(sizeof(float) * ne);
		//}

		int dev = 0;
		hipDeviceProp_t deviceprop;
		CHECK(hipGetDeviceProperties(&deviceprop, dev));
		CHECK(hipSetDevice(dev));

		float* d_sysmat;
		float* d_matfrac;
		float* d_matatn;
		////float* d_tmp;
		//float* d_res1;
		//float* d_res2;
		float* d_res;


		CHECK(hipMalloc((void**)&d_sysmat, nBytes_sysmat));
		CHECK(hipMalloc((void**)&d_matfrac, nBytes_matfrac));
		CHECK(hipMalloc((void**)&d_matatn, nBytes_matatn));
		CHECK(hipMalloc((void**)&d_res, nBytes_res));
		//CHECK(hipMalloc((void**)&d_res1, nBytes_res1));
		//CHECK(hipMalloc((void**)&d_res2, nBytes_res2));

		CHECK(hipMemcpy(d_sysmat, _sysmat, nBytes_sysmat, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_matfrac, _matfrac, nBytes_matfrac, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_matatn, _matatn, nBytes_matatn, hipMemcpyHostToDevice));

		int dimx = 32;
		int dimy = 32;
		dim3 block(dimx, dimy);
		dim3 grid((ne + block.x - 1) / block.x, (ni + block.y - 1) / block.y);
		//im3 grid1((ne + block.x - 1) / block.x, ((nd * nd) + block.y - 1) / block.y);
		//dim3 grid2((ne + block.x - 1) / block.x, (ni + block.y - 1) / block.y);

		//lintg[i][e] += smr[i][j] * matfrac[j][k] * matatn[k][e];

		lintg <<<grid, block>>> (d_sysmat, d_matfrac, d_matatn, d_res, nd, ni, ne, nm);


		//product << < grid1, block >> > (d_matfrac, d_matatn, d_res1, nm, nd*nd, ne);
		//

		////for (int i = 0; i < nd * nd * ne; i+=1) {
		////	printf("%f,", res_test[i]);
		////}

		////lintg << <grid, block >> > (d_sysmat, d_matfrac, d_matatn, d_res, nd, ni, ne, nm);
		//CHECK(hipDeviceSynchronize());
		//CHECK(hipGetLastError());
		////CHECK(hipMemcpy(res_tmp, d_res1, nBytes_res1, hipMemcpyDeviceToHost));

		//product << < grid2, block >> > (d_sysmat, d_res1, d_res2, nd * nd, ni, ne);
		CHECK(hipDeviceSynchronize());
		CHECK(hipGetLastError());
		CHECK(hipMemcpy(res2, d_res, nBytes_res, hipMemcpyDeviceToHost));

		CHECK(hipFree(d_sysmat));
		CHECK(hipFree(d_matfrac));
		CHECK(hipFree(d_matatn));
		CHECK(hipFree(d_res));
		//CHECK(hipFree(d_res1));
		//CHECK(hipFree(d_res2));
		CHECK(hipDeviceReset());

		printf("\n 2nd \n");

		for (int i = 0; i < ni; i++) {
			for (int j = 0; j < ne; j++) {
				res[i][j] = res2[i * ne + j];
				//if (res2[i] != 0) printf("%f,", res2[i]);
			}
		}

		free(_sysmat);
		free(_matfrac);
		free(_matatn);
		free(res2);
		free(res_tmp);

		//return res;
	}

	void calc_gradq(float** res, float** sysmat, float* n, float** nbar, float*** grad_n, int nd, int ni, int nb, int nm, int v_begin, float pixsize) {

		int nBytes_sysmat = sizeof(float) * ni * nd * nd;
		int nBytes_n = sizeof(float) * ni * nb;
		int nBytes_gradn = sizeof(float) * ni * nb * nm;
		int nBytes_res1 = sizeof(float) * nd * nd * nm; // nd*nd X nm

		//for (int j = 0; j < nd * nd; j++) {
		//	//std::cout << "\r" << j << "/" << nd * nd;
		//	for (int k = 0; k < nm; k++) {
		//		for (int i = 0; i < block_size; i++) {
		//			for (int b = 0; b < nb; b++) {
		//				grad_q[j][k] += -smr[i][j] * (1 - (n[i][b] / nbar[i][b])) * grad_n[b][k];
		//			}
		//		}
		//	}
		//}

		float* _sysmat = (float*)malloc(nBytes_sysmat);
		float* _n = (float*)malloc(nBytes_n);
		float* _nbar = (float*)malloc(nBytes_n);
		float* _gradn = (float*)malloc(nBytes_gradn);
		float* res1 = (float*)malloc(nBytes_res1);

		for (int i = 0; i < ni; i++) {
			for (int j = 0; j < nb; j++) {
				_n[(i * nb) + j] = n[(i + v_begin) + j * ni];
				_nbar[(i * nb) + j] = nbar[i][j];
				for (int k = 0; k < nm; k++) {
					_gradn[(i * nb * nm) + (j * nm) + k] = grad_n[i][j][k];
				}
			}
			for (int j = 0; j < nd * nd; j++) {
				_sysmat[(i * nd * nd) + j] = sysmat[i][j] * pixsize;
				/*if (sysmat[i][j] != 0) printf("\nhoge");*/
			}

		}
		//for (int i = 0; i < nb; i++) {
		//	for (int j = 0; j < nm; j++) {

		//		_gradn[(i * nm) + j] = grad_n[i][j];
		//	}
		//}


		//float** res = (float**)malloc(sizeof(float*) * nd * nd); //nd*nd X nm 
		//for (int i = 0; i < nd * nd; i++) {
		//	res[i] = (float*)malloc(sizeof(float) * nm);
		//}

		int dev = 0;
		hipDeviceProp_t deviceprop;
		CHECK(hipGetDeviceProperties(&deviceprop, dev));
		CHECK(hipSetDevice(dev));

		float* d_sysmat;
		float* d_n;
		float* d_nbar;
		float* d_gradn;
		float* d_res1;


		CHECK(hipMalloc((void**)&d_sysmat, nBytes_sysmat));
		CHECK(hipMalloc((void**)&d_n, nBytes_n));
		CHECK(hipMalloc((void**)&d_nbar, nBytes_n));
		CHECK(hipMalloc((void**)&d_gradn, nBytes_gradn));
		CHECK(hipMalloc((void**)&d_res1, nBytes_res1));

		CHECK(hipMemcpy(d_sysmat, _sysmat, nBytes_sysmat, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_n, _n, nBytes_n, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_nbar, _nbar, nBytes_n, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_gradn, _gradn, nBytes_gradn, hipMemcpyHostToDevice));

		int dimx = 32;
		int dimy = 32;
		dim3 block(dimx, dimy);
		dim3 grid((nm + block.x - 1) / block.x, ((nd * nd) + block.y - 1) / block.y);

		//h[j][k][m] += smr[i][j] * (n[i][b] / nbar[i][b]) * suma[i] * source[v_begin + i][b][e] * matatn[k][e] * matatn[m][e] * lintg[i][e];

		//res1[i][b][k][m]


		gradq << <grid, block >> > (d_sysmat, d_n, d_nbar, d_gradn, d_res1, nd, ni, nb, nm);
		CHECK(hipDeviceSynchronize());
		CHECK(hipGetLastError());
		CHECK(hipMemcpy(res1, d_res1, nBytes_res1, hipMemcpyDeviceToHost));

		CHECK(hipFree(d_sysmat));
		CHECK(hipFree(d_n));
		CHECK(hipFree(d_nbar));
		CHECK(hipFree(d_gradn));
		CHECK(hipFree(d_res1));
		CHECK(hipDeviceReset());

		for (int i = 0; i < ni; i++) {
			for (int j = 0; j < nb; j++) {
				if(i % 100 == 0) printf("\nn,nbar, (n/nbar) = %f, %f, %f", _n[i * nb + j], nbar[i][j], _n[i * nb + j] / nbar[i][j]);
			}
		}

		//for (int i = 0; i < ni; i++) {
		//	for (int j = 0; j < nd * nd; j++) {
		//		if(sysmat[i][j] != 0) printf("\nsmr(%d, %d) = %f", i, j, sysmat[i][j]);
		//	}
		//}

		for (int i = 0; i < nd * nd; i++) {
			for (int j = 0; j < nm; j++) {
				res[i][j] = res1[i * nm + j];
				//if (res[i][j] != 0 && i % 1000 == 0) {
				//	printf("at(j,k) = %d, %d, value = %f \n", i, j, res[i][j]);
				//}
				//printf("%f,", res[i][j]);
			}
		}

		free(_sysmat);
		free(_n);
		free(_nbar);
		free(_gradn);
		free(res1);

		//return res;

	}

	void calc_gradq2(float** res, float** sysmat, float*** source, float* n, float** nbar, float** matatn, float** lintg, float* suma, int nd, int ni, int nb, int ne, int nm, int v_begin, float pixsize) {

		int nBytes_source = sizeof(float) * ni * nb * ne;
		int nBytes_sysmat = sizeof(float) * ni * nd * nd;
		int nBytes_n = sizeof(float) * ni * nb;
		int nBytes_matatn = sizeof(float) * nm * ne;
		int nBytes_lintg = sizeof(float) * ni * ne;
		int nBytes_suma = sizeof(float) * ni;
		int nBytes_res1 = sizeof(float) * ni * nb * nm; // ni*nb X nm
		int nBytes_res2 = sizeof(float) * nd * nd * nm; // nd*nd X nm

		float* _source = (float*)malloc(nBytes_source);
		float* _sysmat = (float*)malloc(nBytes_sysmat);
		float* _n = (float*)malloc(nBytes_n);
		float* _nbar = (float*)malloc(nBytes_n);
		float* _matatn = (float*)malloc(nBytes_matatn);
		float* _lintg = (float*)malloc(nBytes_lintg);
		float* res1 = (float*)malloc(nBytes_res1);
		float* res2 = (float*)malloc(nBytes_res2);

		for (int i = 0; i < ni; i++) {
			for (int j = 0; j < nb; j++) {
				for (int k = 0; k < ne; k++) {
					_source[(i * nb * ne) + (j * ne) + k] = source[i + v_begin][j][k];
				}
				_n[(i * nb) + j] = n[(i + v_begin) + j * ni];
				_nbar[(i * nb) + j] = nbar[i][j];
			}
			for (int j = 0; j < ne; j++) {
				_lintg[(i * ne) + j] = lintg[i][j];
			}
			for (int j = 0; j < nd * nd; j++) {
				_sysmat[(i * nd * nd) + j] = sysmat[i][j] * pixsize;
			}
		}
		for (int i = 0; i < nm; i++) {
			for (int j = 0; j < ne; j++) {
				_matatn[(i * ne) + j] = matatn[i][j];
			}
		}

		int dev = 0;
		hipDeviceProp_t deviceprop;
		CHECK(hipGetDeviceProperties(&deviceprop, dev));
		CHECK(hipSetDevice(dev));

		float* d_source;
		float* d_sysmat;
		float* d_n;
		float* d_nbar;
		float* d_matatn;
		float* d_lintg;
		float* d_suma;
		float* d_res1;
		float* d_res2;


		CHECK(hipMalloc((void**)&d_source, nBytes_source));
		CHECK(hipMalloc((void**)&d_sysmat, nBytes_sysmat));
		CHECK(hipMalloc((void**)&d_n, nBytes_n));
		CHECK(hipMalloc((void**)&d_nbar, nBytes_n));
		CHECK(hipMalloc((void**)&d_matatn, nBytes_matatn));
		CHECK(hipMalloc((void**)&d_lintg, nBytes_lintg));
		CHECK(hipMalloc((void**)&d_suma, nBytes_suma));
		CHECK(hipMalloc((void**)&d_res1, nBytes_res1));
		CHECK(hipMalloc((void**)&d_res2, nBytes_res2));

		CHECK(hipMemcpy(d_source, _source, nBytes_source, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_sysmat, _sysmat, nBytes_sysmat, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_n, _n, nBytes_n, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_nbar, _nbar, nBytes_n, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_matatn, _matatn, nBytes_matatn, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_lintg, _lintg, nBytes_lintg, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_suma, suma, nBytes_suma, hipMemcpyHostToDevice));

		int dimx = 32;
		int dimy = 32;
		dim3 block(dimx, dimy);
		dim3 grid1((nm + block.x - 1) / block.x, ((ni * nb) + block.y - 1) / block.y);
		dim3 grid2((nm + block.x - 1) / block.x, ((nd * nd) + block.y - 1) / block.y);

		gradq2_1 << <grid1, block >> > (d_source, d_matatn, d_lintg, d_res1, ni, nb, ne, nm);
		CHECK(hipDeviceSynchronize());
		CHECK(hipGetLastError());

		printf("hogehoge");

		gradq2_2 << <grid2, block >> > (d_sysmat, d_n, d_nbar, d_suma, d_res1, d_res2, nd, ni, nb, nm);
		CHECK(hipDeviceSynchronize());
		CHECK(hipGetLastError());
		CHECK(hipMemcpy(res2, d_res2, nBytes_res2, hipMemcpyDeviceToHost));

		printf("piyo");

		CHECK(hipFree(d_source));
		CHECK(hipFree(d_sysmat));
		CHECK(hipFree(d_n));
		CHECK(hipFree(d_nbar));
		CHECK(hipFree(d_matatn));
		CHECK(hipFree(d_lintg));
		CHECK(hipFree(d_suma));
		CHECK(hipFree(d_res1));
		CHECK(hipFree(d_res2));
		CHECK(hipDeviceReset());

		printf("hagehage");

		for (int i = 0; i < nd * nd; i++) {
			for (int j = 0; j < nm; j++) {
				res[i][j] = res2[i * nm + j];
			}
			//if (res[i][0] < -1000) {
			//	printf("\ngradq, n/nbar  %f  %f", res[i][0], n[]);
			//}
		}

		free(_source);
		free(_sysmat);
		free(_n);
		free(_nbar);
		free(_matatn);
		free(_lintg);
		free(res1);
		free(res2);

	}

	float calc_gradp(float* n, float** nbar, float** nbard, int nd, int ni, int nb, int ne, int v_begin) {

		int nBytes_n = sizeof(float) * ni * nb;
		int nBytes_res = sizeof(float) * nd * nd; // nd X nd

		float* _n = (float*)malloc(nBytes_n);
		float* _nbar = (float*)malloc(nBytes_n);
		float* _nbard = (float*)malloc(nBytes_n);
		float* res1 = (float*)malloc(nBytes_res);

		for (int i = 0; i < ni; i++) {
			for (int j = 0; j < nb; j++) {
				_n[(i * nb) + j] = n[(i + v_begin) + j * ni];
				_nbar[(i * nb) + j] = nbar[i][j];
				_nbard[(i * nb) + j] = nbard[i][j];
			}
		}

		int dev = 0;
		hipDeviceProp_t deviceprop;
		CHECK(hipGetDeviceProperties(&deviceprop, dev));
		CHECK(hipSetDevice(dev));

		float* d_n;
		float* d_nbar;
		float* d_nbard;
		float* d_res1;

		CHECK(hipMalloc((void**)&d_n, nBytes_n));
		CHECK(hipMalloc((void**)&d_nbar, nBytes_n));
		CHECK(hipMalloc((void**)&d_nbard, nBytes_n));

		CHECK(hipMalloc((void**)&d_res1, nBytes_res));

		CHECK(hipMemcpy(d_n, _n, nBytes_n, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_nbar, _nbar, nBytes_n, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_nbard, _nbard, nBytes_n, hipMemcpyHostToDevice));

		int dimx = 32;
		int dimy = 32;
		dim3 block(dimx, dimy);
		dim3 grid((nd + block.x - 1) / block.x, (nd + block.y - 1) / block.y);

		gradp << <grid, block >> > (d_n, d_nbar, d_nbard, d_res1, nd, ni, nb);
		CHECK(hipDeviceSynchronize());
		CHECK(hipGetLastError());
		CHECK(hipMemcpy(res1, d_res1, nBytes_res, hipMemcpyDeviceToHost));

		printf("piyo");


		CHECK(hipFree(d_n));
		CHECK(hipFree(d_nbar));		
		CHECK(hipFree(d_nbard));
		CHECK(hipFree(d_res1));
		CHECK(hipDeviceReset());

		printf("hagehage");


		float res = 0;// res2[0];

		for (int j = 0; j < nd * nd; j++) {
			res += res1[j];
		}

		free(_n);
		free(_nbar);
		free(_nbard);
		free(res1);

		return res;

	}

	void calc_hesseq(float*** res, float*** source, float** sysmat, float* n, float** nbar, float** matatn, float** lintg, float* suma, int nd, int ni, int nb, int ne, int nm, int v_begin, float pixsize) {

		int nBytes_source = sizeof(float) * ni * nb * ne;
		int nBytes_sysmat = sizeof(float) * ni * nd * nd;
		int nBytes_n = sizeof(float) * ni * nb;
		int nBytes_matatn = sizeof(float) * nm * ne;
		int nBytes_lintg = sizeof(float) * ni * ne;
		int nBytes_suma = sizeof(float) * ni;
		int nBytes_res1 = sizeof(float) * ni * nb * nm * nm; // ni*nb X nm*nm
		int nBytes_res2 = sizeof(float) * nd * nd * nm * nm;

		float* _source = (float*)malloc(nBytes_source);
		float* _sysmat = (float*)malloc(nBytes_sysmat);
		float* _n      = (float*)malloc(nBytes_n);
		float* _nbar   = (float*)malloc(nBytes_n);
		float* _matatn = (float*)malloc(nBytes_matatn);
		float* _lintg  = (float*)malloc(nBytes_lintg);
		//float* res1    = (float*)malloc(nBytes_res1);
		float* res2    = (float*)malloc(nBytes_res2);

		for (int i = 0; i < ni; i++) {
			for (int j = 0; j < nb; j++) {
				for (int k = 0; k < ne; k++) {
					_source[(i * nb * ne) + (j * ne) + k] = source[i + v_begin][j][k];
				}
				_n[(i * nb) + j] = n[(i + v_begin) + j * ni];
				_nbar[(i * nb) + j] = nbar[i][j];
			}
			for (int j = 0; j < nd * nd; j++) {
				_sysmat[(i * nd * nd) + j] = sysmat[i][j] * pixsize;
			}
			for (int j = 0; j < ne; j++) {
				_lintg[(i * ne) + j] = lintg[i][j];
			}
		}
		for (int i = 0; i < nm; i++) {
			for (int j = 0; j < ne; j++) {
				_matatn[(i * ne) + j] = matatn[i][j];
			}
		}


		//float*** res_tmp = (float***)malloc(sizeof(float**) * ni * nb); //ni*nb X nm X nm
		//for (int i = 0; i < ni * nb; i++) {
		//	res_tmp[i] = (float**)malloc(sizeof(float*) * nm);
		//	for (int j = 0; j < nm; j++) {
		//		res_tmp[i][j] = (float*)malloc(sizeof(float) * nm);
		//	}
		//}


		int dev = 0;
		hipDeviceProp_t deviceprop;
		CHECK(hipGetDeviceProperties(&deviceprop, dev));
		CHECK(hipSetDevice(dev));


		float* d_source;
		float* d_sysmat;
		float* d_n;
		float* d_nbar;
		float* d_matatn;
		float* d_lintg;
		float* d_suma;
		float* d_res1;
		float* d_res2;


		CHECK(hipMalloc((void**)&d_source, nBytes_source));
		CHECK(hipMalloc((void**)&d_sysmat, nBytes_sysmat));
		CHECK(hipMalloc((void**)&d_n, nBytes_n));
		CHECK(hipMalloc((void**)&d_nbar, nBytes_n));
		CHECK(hipMalloc((void**)&d_matatn, nBytes_matatn));
		CHECK(hipMalloc((void**)&d_lintg, nBytes_lintg));
		CHECK(hipMalloc((void**)&d_suma, nBytes_suma));
		CHECK(hipMalloc((void**)&d_res1, nBytes_res1));
		CHECK(hipMalloc((void**)&d_res2, nBytes_res2));

		CHECK(hipMemcpy(d_source, _source, nBytes_source, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_sysmat, _sysmat, nBytes_sysmat, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_n, _n, nBytes_n, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_nbar, _nbar, nBytes_n, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_matatn, _matatn, nBytes_matatn, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_lintg, _lintg, nBytes_lintg, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_suma, suma, nBytes_suma, hipMemcpyHostToDevice));

		int dimx = 32;
		int dimy = 32;
		dim3 block(dimx, dimy);
		dim3 grid1(((nm * nm) + block.x - 1) / block.x, ((ni * nb) + block.y - 1) / block.y);
		dim3 grid2(((nm * nm) + block.x - 1) / block.x, ((nd * nd) + block.y - 1) / block.y);

		//h[j][k][m] += smr[i][j] * (n[i][b] / nbar[i][b]) * suma[i] * source[v_begin + i][b][e] * matatn[k][e] * matatn[m][e] * lintg[i][e];

		//res1[i][b][k][m]

		hesseq1 << <grid1, block >> > (d_source, d_matatn, d_lintg, d_res1, ni, nb, ne, nm); // res1: ni*nb X nm*nm
		CHECK(hipDeviceSynchronize());
		CHECK(hipGetLastError());

		//res2 = h[j][k][m]
		//CHECK(hipMemcpy(res1, d_res1, nBytes_res1, hipMemcpyDeviceToHost));

		int count = 0;
		//for (int i = 0; i < ni * nb; i++) {
		//	for (int j = 0; j < nm; j++) {
		//		for (int k = 0; k < nm; k++) {
		//			res_tmp[i][j][k] = res1[i * nm * nm + j * nm + k];
		//			if (j != k && res_tmp[i][j][k] != 0) {
		//				count++;
		//			}
		//		}
		//	}
		//}
		//printf("not 0 count: %d / %d", count, ni * nb * nm * nm);


		hesseq2 << <grid2, block >> > (d_sysmat, d_n, d_nbar, d_suma, d_res1, d_res2, nd, ni, nb, nm);
		CHECK(hipDeviceSynchronize());
		CHECK(hipGetLastError());
		CHECK(hipMemcpy(res2, d_res2, nBytes_res2, hipMemcpyDeviceToHost));

		//res[idx] += sysmat[i * nd * nd + iy] * (n[i * nb + b] / nbar[i * nb + b]) * suma[i] * tmp[i * nb * nm * nm + b * nm * nm + ix];

		/*hesseq << <grid, block >> > (d_source, d_sysmat, d_n, d_nbar, d_matatn, d_lintg, d_suma, d_res, nd, ni, nb, ne, nm); 
		CHECK(hipDeviceSynchronize());
		CHECK(hipGetLastError());
		CHECK(hipMemcpy(res1, d_res, nBytes_res, hipMemcpyDeviceToHost));*/

		CHECK(hipFree(d_source));
		CHECK(hipFree(d_sysmat));
		CHECK(hipFree(d_n));
		CHECK(hipFree(d_nbar));
		CHECK(hipFree(d_matatn));
		CHECK(hipFree(d_lintg));
		CHECK(hipFree(d_suma));
		CHECK(hipFree(d_res1));
		CHECK(hipFree(d_res2));
		CHECK(hipDeviceReset());

		count = 0;

		for (int i = 0; i < nd * nd; i++) {
			for (int j = 0; j < nm; j++) {
				for (int k = 0; k < nm; k++) {
					res[i][j][k] = res2[i * nm * nm + j * nm + k];
					if (j != k && res[i][j][k] != 0) {
						count++;
					}
				}
			}
		}

		printf("(2)not 0 count: %d / %d", count, nd * nd * nm * nm);

		free(_source);
		free(_sysmat);
		free(_n);
		free(_nbar);
		free(_matatn);
		free(_lintg);
		//free(res1);
		free(res2);

		//return res;

	}

	void calc_hesseqp(float* res, float*** source, float** nbar, float** nbard, float** matatn, float** lintg, float* suma, int nd, int ni, int nb, int ne, int nm, int v_begin) {

		int nBytes_source = sizeof(float) * ni * nb * ne;
		int nBytes_n = sizeof(float) * ni * nb;
		int nBytes_matatn = sizeof(float) * nm * ne;
		int nBytes_lintg = sizeof(float) * ni * ne;
		int nBytes_suma = sizeof(float) * ni;
		int nBytes_res1 = sizeof(float) * ni * nb * nm; // ni*nb X nm*nm
		//int nBytes_res2 = sizeof(float) * nd * nd * nm;

		float* _source = (float*)malloc(nBytes_source);
		float* _nbar = (float*)malloc(nBytes_n);
		float* _nbard = (float*)malloc(nBytes_n);
		float* _matatn = (float*)malloc(nBytes_matatn);
		float* _lintg = (float*)malloc(nBytes_lintg);
		float* res1 = (float*)malloc(nBytes_res1);
		//float* res2 = (float*)malloc(nBytes_res2);

		for (int i = 0; i < ni; i++) {
			for (int j = 0; j < nb; j++) {
				for (int k = 0; k < ne; k++) {
					_source[(i * nb * ne) + (j * ne) + k] = source[i + v_begin][j][k];
				}
				_nbar[(i * nb) + j] = nbar[i][j];
				_nbard[(i * nb) + j] = nbard[i][j];
			}
			for (int j = 0; j < ne; j++) {
				_lintg[(i * ne) + j] = lintg[i][j];
			}
		}
		for (int i = 0; i < nm; i++) {
			for (int j = 0; j < ne; j++) {
				_matatn[(i * ne) + j] = matatn[i][j];
			}
		}


		int dev = 0;
		hipDeviceProp_t deviceprop;
		CHECK(hipGetDeviceProperties(&deviceprop, dev));
		CHECK(hipSetDevice(dev));


		float* d_source;
		float* d_nbar;
		float* d_nbard;
		float* d_matatn;
		float* d_lintg;
		float* d_suma;
		float* d_res1;
		//float* d_res2;


		CHECK(hipMalloc((void**)&d_source, nBytes_source));
		CHECK(hipMalloc((void**)&d_nbar, nBytes_n));
		CHECK(hipMalloc((void**)&d_nbard, nBytes_n));
		CHECK(hipMalloc((void**)&d_matatn, nBytes_matatn));
		CHECK(hipMalloc((void**)&d_lintg, nBytes_lintg));
		CHECK(hipMalloc((void**)&d_suma, nBytes_suma));
		CHECK(hipMalloc((void**)&d_res1, nBytes_res1));
		//CHECK(hipMalloc((void**)&d_res2, nBytes_res2));

		CHECK(hipMemcpy(d_source, _source, nBytes_source, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_nbar, _nbar, nBytes_n, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_nbard, _nbar, nBytes_n, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_matatn, _matatn, nBytes_matatn, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_lintg, _lintg, nBytes_lintg, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_suma, suma, nBytes_suma, hipMemcpyHostToDevice));

		int dimx = 32;
		int dimy = 32;
		dim3 block(dimx, dimy);
		dim3 grid((nm + block.x - 1) / block.x, ((ni * nb) + block.y - 1) / block.y);
		//dim3 grid2((nm + block.x - 1) / block.x, ((nd * nd) + block.y - 1) / block.y);


		hesseqp << <grid, block >> > (d_source, d_matatn, d_lintg, d_res1, ni, nb, ne, nm); // res1: ni*nb X nm
		CHECK(hipDeviceSynchronize());
		CHECK(hipGetLastError());
		CHECK(hipMemcpy(res1, d_res1, nBytes_res1, hipMemcpyDeviceToHost));

		for (int m = 0; m < nm; m++) {
			for (int i = 0; i < ni; i++) {
				for (int b = 0; b < nb; b++) {
					res[m] += suma[i] * (nbard[i][b] / nbar[i][b]) * res1[i * nb * nm + b * nm + m] * 0.5;
					//res[m] = 0;
				}
			}
		}



		//int count = 0;

		//hesseqp2 << <grid2, block >> > (d_nbar, d_nbard, d_suma, d_res1, d_res2, nd, ni, nb, nm); // res2: nd*nd X nm
		//CHECK(hipDeviceSynchronize());
		//CHECK(hipGetLastError());
		//CHECK(hipMemcpy(res2, d_res2, nBytes_res2, hipMemcpyDeviceToHost));

		//__global__ void hesseqp2(float* nbar, float* nbard, float* suma, float* tmp, float* res, const int nd, const int ni, const int nb, const int nm) {
		//	unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
		//	unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
		//	unsigned int idx = iy * nm + ix; // nd*nd X nm

		//	if (ix < nm && iy < nd * nd) {
		//		res[idx] = 0;
		//		for (int i = 0; i < ni; i++) {
		//			for (int b = 0; b < nb; b++) {
		//				res[idx] += suma[i] * (nbard[i * nb + b] / nbar[i * nb + b]) * tmp[i * nb * nm + b * nm + ix] * 0.5;
		//			}
		//		}
		//	}
		//}


		CHECK(hipFree(d_source));
		CHECK(hipFree(d_nbar));
		CHECK(hipFree(d_nbard));
		CHECK(hipFree(d_matatn));
		CHECK(hipFree(d_lintg));
		CHECK(hipFree(d_suma));
		CHECK(hipFree(d_res1));
		//CHECK(hipFree(d_res2));
		CHECK(hipDeviceReset());

		//count = 0;

		//for (int i = 0; i < nd * nd; i++) {
		//	for (int j = 0; j < nm; j++) {
		//		res[i][j] = res2[i * nm + j];
		//	}
		//}

		//for (int i = 0; i < nd * nd; i++) {
		//	for (int j = 0; j < nm; j++) {
		//		res[i] += res2[i * nm + j];
		//	}
		//}


		free(_source);
		free(_nbar);
		free(_nbard);
		free(_matatn);
		free(_lintg);
		free(res1);
		//free(res2);

	}

	float calc_hessep(float* n, float** nbar, float** nbard, int ni, int nb) {

		float res = 0;
		float tmp;
		for (int i = 0; i < ni; i++) {
			for (int b = 0; b < nb; b++) {
				tmp = nbard[i][b] / nbar[i][b];
				res += n[i * nb + b] * tmp * tmp + (1 - n[i * nb + b] / nbar[i][b]) * nbard[i][b] * 0.5;
			}
		}

		return res;

		//int nBytes_source = sizeof(float) * ni * nb * ne;
		//int nBytes_n = sizeof(float) * ni * nb;
		//int nBytes_matatn = sizeof(float) * nm * ne;
		//int nBytes_lintg = sizeof(float) * ni * ne;
		//int nBytes_suma = sizeof(float) * ni;
		//int nBytes_res1 = sizeof(float) * ni * nb * nm; // ni*nb X nm*nm
		//int nBytes_res2 = sizeof(float) * nd * nd * nm;

		//float* _source = (float*)malloc(nBytes_source);
		//float* _nbar = (float*)malloc(nBytes_n);
		//float* _nbard = (float*)malloc(nBytes_n);
		//float* _matatn = (float*)malloc(nBytes_matatn);
		//float* _lintg = (float*)malloc(nBytes_lintg);
		//float* res2 = (float*)malloc(nBytes_res2);

		//for (int i = 0; i < ni; i++) {
		//	for (int j = 0; j < nb; j++) {
		//		for (int k = 0; k < ne; k++) {
		//			_source[(i * nb * ne) + (j * ne) + k] = source[i + v_begin][j][k];
		//		}
		//		_nbar[(i * nb) + j] = nbar[i][j];
		//		_nbard[(i * nb) + j] = nbard[i][j];
		//	}
		//	for (int j = 0; j < ne; j++) {
		//		_lintg[(i * ne) + j] = lintg[i][j];
		//	}
		//}
		//for (int i = 0; i < nm; i++) {
		//	for (int j = 0; j < ne; j++) {
		//		_matatn[(i * ne) + j] = matatn[i][j];
		//	}
		//}


		//int dev = 0;
		//hipDeviceProp_t deviceprop;
		//CHECK(hipGetDeviceProperties(&deviceprop, dev));
		//CHECK(hipSetDevice(dev));


		//float* d_source;
		//float* d_nbar;
		//float* d_nbard;
		//float* d_matatn;
		//float* d_lintg;
		//float* d_suma;
		//float* d_res1;
		//float* d_res2;


		//CHECK(hipMalloc((void**)&d_source, nBytes_source));
		//CHECK(hipMalloc((void**)&d_nbar, nBytes_n));
		//CHECK(hipMalloc((void**)&d_nbard, nBytes_n));
		//CHECK(hipMalloc((void**)&d_matatn, nBytes_matatn));
		//CHECK(hipMalloc((void**)&d_lintg, nBytes_lintg));
		//CHECK(hipMalloc((void**)&d_suma, nBytes_suma));
		//CHECK(hipMalloc((void**)&d_res1, nBytes_res1));
		//CHECK(hipMalloc((void**)&d_res2, nBytes_res2));

		//CHECK(hipMemcpy(d_source, _source, nBytes_source, hipMemcpyHostToDevice));
		//CHECK(hipMemcpy(d_nbar, _nbar, nBytes_n, hipMemcpyHostToDevice));
		//CHECK(hipMemcpy(d_nbard, _nbar, nBytes_n, hipMemcpyHostToDevice));
		//CHECK(hipMemcpy(d_matatn, _matatn, nBytes_matatn, hipMemcpyHostToDevice));
		//CHECK(hipMemcpy(d_lintg, _lintg, nBytes_lintg, hipMemcpyHostToDevice));
		//CHECK(hipMemcpy(d_suma, suma, nBytes_suma, hipMemcpyHostToDevice));

		//int dimx = 32;
		//int dimy = 32;
		//dim3 block(dimx, dimy);
		//dim3 grid1((nm + block.x - 1) / block.x, ((ni * nb) + block.y - 1) / block.y);
		//dim3 grid2((nm + block.x - 1) / block.x, ((nd * nd) + block.y - 1) / block.y);


		//hessep << <grid, block >> > (d_nbar, d_nbard, d_suma, d_res1, d_res2, nd, ni, nb, nm); // res2: nd*nd X nm
		//CHECK(hipDeviceSynchronize());
		//CHECK(hipGetLastError());
		//CHECK(hipMemcpy(res2, d_res2, nBytes_res2, hipMemcpyDeviceToHost));


		//CHECK(hipFree(d_source));
		//CHECK(hipFree(d_nbar));
		//CHECK(hipFree(d_nbard));
		//CHECK(hipFree(d_matatn));
		//CHECK(hipFree(d_lintg));
		//CHECK(hipFree(d_suma));
		//CHECK(hipFree(d_res1));
		//CHECK(hipFree(d_res2));
		//CHECK(hipDeviceReset());

		//count = 0;

		//for (int i = 0; i < nd * nd; i++) {
		//	for (int j = 0; j < nm; j++) {
		//		res[i][j] = res2[i * nm + j];
		//	}
		//}


		//free(_source);
		//free(_nbar);
		//free(_nbard);
		//free(_matatn);
		//free(_lintg);
		////free(res1);
		//free(res2);
	}

}
