#include "hip/hip_runtime.h"
#include "CalcImgdiff.cuh"


#define CHECK(call)\
{\
    const hipError_t error = call;\
    if (error != hipSuccess)\
        {\
            printf("Error: %s:%d",__FILE__,__LINE__);\
            printf("code:%d, reason: %s\n", error, hipGetErrorString(error));\
            exit(1);\
        }\
}

/*

namespace Reconstruction {

	__global__ void calc_imgdiff(float int row, int num_col,) {

	}

	void CalcImgdiff(float* attenu, float* imgdiff, const float* sino, const float* elem, const float* rowptr, const float* colind, int blocksize, int nd, int nv, int ne)
	{


		float* _sysmatrow = (float*)malloc(sizeof(float) * nd * nd);

		bool block = true;
		bool init = true;

		bool eachblockmode = false;

		for (int j = 0; j < nd * nd; j++) {
			imgdiff[j] = 0;
		}

		calc_imgdiff();

		for (int j = 0; j < blocksize; j++) {
			calc_imgdiff();
			//(*_sysmatblock).Extract_row_dense(j, nd * nd, _sysmatrow);
			calc_imgdiff(imgdiff, _sysmatrow, attenu, sino[i * blocksize + j], nd * nd);
			Reconstruction::mul_array1(imgdiff, nd, nd * nd);
		}
	}

	void Extract_row_dense(int row, int num_col, float* vec) {

		for (int i = 0; i < num_col; i++) {
			vec[i] = 0;
		}

		for (int i = rowptr.get()[row]; i < rowptr.get()[row + 1]; i++) {
			int tmp = colind.get()[i];
			vec[tmp] = elements.get()[i];
		}

	}

	void IterationRec::calc_imgdiff(float* idiff, float* smr, float* atn, float sn, int size) const {
		float sys_atn = Reconstruction::dot_array(smr, atn, size);
		float sys_sys = Reconstruction::dot_array(smr, smr, size);

		Reconstruction::mul_array1(smr, ((sys_atn - sn) / sys_sys), size);
		Reconstruction::add_array(idiff, smr, size);
	}
}

*/